#include "hip/hip_runtime.h"
# include "../common.h"
using namespace std;
# include "../lsoda/cuLsoda_all.cu"
# include "../lsoda/cuLsoda.hpp"
# include "./opt.hpp"
// A guassian distribution with mean 0 standard deviation of 1
double guassrand()
{
      static double V1, V2, S;
      static int phase = 0;
      double X;

      srand(time(NULL));
      if (phase == 0) 
      {
      	 do {
	    double U1 = (double)rand() / RAND_MAX;
	    double U2 = (double)rand() / RAND_MAX;
            
            V1 = 2 * U1 - 1;
	    V2 = 2 * U2 - 1;
	    S = V1 * V1 + V2 * V2;
	    } while(S >= 1 || S == 0);

	    X = V1 * sqrt(-2 * log(S) / S);
      } else
      X = V2 * sqrt(-2 * log(S) / S);
      
      phase = 1 - phase;

      return X;
}
// LSODE integrator function
//void integrate_lsoda_ode ( const vector <double> &x_d, const vector <double> &t_d, const vector <double> &sd_d, const myFex &fex, const myJex &jex, state_type &error_sim_d)
void integrate_lsoda_ode ( const int &gene_ind, const vector <double> &x_d, const vector <double> &t_d, const double &mean_xd, const myFex &fex, const myJex &jex, state_type &error_sim_d)
{
   bool check_int = false; 
   host_type error_sim_h(probSize);
   //state_type error_sim_d(probSize);
   // Studying gene 20 for the N= 25 set
   /* Local variables: input arguments for Lsoda.
      For a more detailed description see cuLsoda.cu L 171*/
   // initial value of independent variable t
   double *t = (double*)malloc(sizeof(double)*probSize);
   // initial value of dependent variable Y.SIZE() = NEQ 
   double *y = (double*)malloc(sizeof(double)*probSize);
   // JT is the jacobian type indicator
   int *jt = (int*)malloc(sizeof(int)*probSize);
   // NEQ is the number of equations
   int *neq = (int*)malloc(sizeof(int)*probSize);
   // lengths of IWORK and RWORK
   int *liw = (int*)malloc(sizeof(int)*probSize);
   int *lrw = (int*)malloc(sizeof(int)*probSize);
   // ATOL is the absolute tolerance parameter
   double *atol = (double*)malloc(sizeof(double)*probSize);
   // ITOL size of ATOL, can be same or different for each EQ in NEQ
   int *itol =(int*) malloc(sizeof(int)*probSize);
   // IOPT optional inputs argument
   int *iopt =(int*) malloc(sizeof(int)*probSize);
   // RTOL relative tolerance parameter
   double *rtol = (double*)malloc(sizeof(double)*probSize);
   // IOUT forward step iterations   
   //int *iout =(int*) malloc(sizeof(int)*probSize);
   // TOUT time point where output is desired    
   double *tout =(double*) malloc(sizeof(double)*probSize);
   // ITASK determines normal computation of Y at TOUT
   int *itask = (int*)malloc(sizeof(int)*probSize);
   // IWORK int array of length of at least 20 + NEQ
   int *iwork =(int*) malloc(sizeof(int)*21*probSize);
   // RWORK array of length of at least 22 + NEQ * MAX(16, NEQ + 9)
   double *rwork = (double*)malloc(sizeof(double)*38*probSize);
   // ISTATE input flag  
   int *istate = (int*)malloc(sizeof(int)*probSize);
   // COMMON BLOCK DECLARATIONS
   struct cuLsodaCommonBlock common[probSize];
   struct cuLsodaCommonBlock *Hcommon = common;
   int *err = (int*)malloc(sizeof(int)*probSize);
   //  End Local Block 

   // Pointers to Device versions of Local variables 
   double *_Dt;
   double *_Dy;	// [3]
   int *_Djt;
   int *_Dneq;
   int *_Dliw;
   int *_Dlrw;
   double *_Datol;	//[3]
   int *_Ditol;
   int *_Diopt;
   double *_Drtol;
   double *_Dtout;
   int *_Ditask;
   int *_Diwork;	// [23]
   double *_Drwork;	// [70]
   int *_Distate;
   struct cuLsodaCommonBlock *_Dcommon;
   int *_Derr;
   // End Pointer Block 
   // Transfer other data to device
   state_type x_d_d = x_d;
   state_type t_d_d = t_d;
   //state_type sd_d_d = sd_d;
   host_type xd_h, td_h;
   //host_type y_val_h(N_time_points*probSize);
	
   // Method instantiations for Derivative and Jacobian functions to send to template 
   thrust::device_ptr<double> t0_ptr;
   thrust::device_ptr<int> index_ptr;
   thrust::fill(error_sim_d.begin(), error_sim_d.end(), 0.0);
   // Assignment of initial values to locals 
   for (int i = 0; i < probSize; i++)
   {
      *(neq+i) = 1;
      *(y+0+i) = x_d[gene_ind*N_time_points];
      //*(y+0+i) = 1.0;
      *(t+i) = (double)0.;
      *(tout+i) = 1.0;
      *(itol+i) = 1;
      *(rtol+i) = (double)1e-3;
      *(atol+i) = (double)1e-6;
      *(itask+i) = 1;
      *(istate+i) = 1;
      *(iopt+i) = 0;
      *(lrw+i) = 38;
      *(liw+i) = 21;
      *(jt+i) = 2;
      cuLsodaCommonBlockInit(&Hcommon[i]);
      *(err +i) = -1;
   } 
   // Allocate device memory for each of the pointers, and copy the values from local to device
   hipMalloc((void**)&_Dt,sizeof(double)*probSize);
   hipMemcpy(_Dt,t,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dy,sizeof(double)*probSize);							
   hipMemcpy(_Dy,y,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Djt,sizeof(int)*probSize);
   hipMemcpy(_Djt,jt,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dneq,sizeof(int)*probSize);
   hipMemcpy(_Dneq,neq,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dliw,sizeof(int)*probSize);
   hipMemcpy(_Dliw,liw,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dlrw,sizeof(int)*probSize);
   hipMemcpy(_Dlrw,lrw,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Datol,sizeof(double)*probSize);
   hipMemcpy(_Datol,atol,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Ditol,sizeof(int)*probSize);							
   hipMemcpy(_Ditol,itol,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Diopt,sizeof(int)*probSize);							
   hipMemcpy(_Diopt,iopt,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Drtol,sizeof(double)*probSize);							
   hipMemcpy(_Drtol,rtol,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dtout,sizeof(double)*probSize);
   hipMemcpy(_Dtout,tout,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Ditask,sizeof(int)*probSize);
   hipMemcpy(_Ditask,itask,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Diwork,sizeof(int)*21*probSize);
   hipMemcpy(_Diwork,iwork,sizeof(int)*21*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Drwork,sizeof(double)*38*probSize);
   hipMemcpy(_Drwork,rwork,sizeof(double)*38*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Distate,sizeof(int)*probSize);							
   hipMemcpy(_Distate,istate,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dcommon,sizeof(struct cuLsodaCommonBlock)*probSize);	
   hipMemcpy(_Dcommon,Hcommon,sizeof(struct cuLsodaCommonBlock)*probSize, hipMemcpyHostToDevice);
   hipMalloc((void**)&_Derr,sizeof(double)*probSize);
   hipMemcpy(_Derr,istate,sizeof(double)*probSize,hipMemcpyHostToDevice);
   // End Allocation and Copy Block 
   thrust::device_ptr<double> y_dev_p = thrust::device_pointer_cast(_Dy);
   thrust::device_ptr<double> t_dev_p = thrust::device_pointer_cast(_Dt);
   thrust::device_ptr<double> tout_dev_p = thrust::device_pointer_cast(_Dtout);
   cuLsoda<<<blocksPerGrid,threadsPerBlock>>>(fex, _Dneq, _Dy, _Dt, _Dtout, _Ditol, _Drtol, _Datol, _Ditask, _Distate, _Diopt, _Drwork, _Dlrw, _Diwork, _Dliw, jex, _Djt, _Dcommon, _Derr, probSize);
   //thrust::copy(y_dev_p,y_dev_p+probSize,y_val_h.begin());
   //xd_h.push_back(y_dev_p[0]);
   //td_h.push_back(tout_dev_p[0]);
   thrust::fill(tout_dev_p, tout_dev_p+probSize, t_d_d[1]);
   thrust::fill(y_dev_p, y_dev_p+probSize, x_d_d[gene_ind*N_time_points]);
   //thrust::fill(y_dev_p, y_dev_p+probSize, 1.0);
   thrust::fill(t_dev_p, t_dev_p+probSize, 0.0);
   //for ( int t_ind = 1; t_ind <= N_time_points; t_ind++)
   int t_ind = 1; 
   while ( t_ind < N_time_points )
   {
       cuLsoda<<<blocksPerGrid,threadsPerBlock>>>(fex, _Dneq, _Dy, _Dt, _Dtout, _Ditol, _Drtol, _Datol, _Ditask, _Distate, _Diopt, _Drwork, _Dlrw, _Diwork, _Dliw, jex, _Djt, _Dcommon, _Derr, probSize);
       if (check_int)
       {
          xd_h.push_back(y_dev_p[0]);
          td_h.push_back(tout_dev_p[0]);
       }
       thrust::fill(tout_dev_p, tout_dev_p+probSize, t_d_d[t_ind+1]);
       thrust::transform(error_sim_d.begin(), error_sim_d.end(), y_dev_p, 
                         error_sim_d.begin(), error_functor(x_d_d[gene_ind*N_time_points+t_ind],mean_xd));
       //thrust::fill(y_dev_p, y_dev_p+probSize, 1.0);
       //thrust::fill(t_dev_p, t_dev_p+probSize, 0.0);
       t_ind++;
   }
   error_sim_h = error_sim_d;
   // Copy memory back from Device to Host 
   hipMemcpy(t,_Dt,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(y,_Dy,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(jt,_Djt,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(neq,_Dneq,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(liw,_Dliw,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(lrw,_Dlrw,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(atol,_Datol,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(itol,_Ditol,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(iopt,_Diopt,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(rtol,_Drtol,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(tout,_Dtout,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(itask,_Ditask,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(iwork,_Diwork,sizeof(int)*21*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(rwork,_Drwork,sizeof(double)*38*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(istate,_Distate,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(Hcommon,_Dcommon,sizeof(struct cuLsodaCommonBlock)*probSize, hipMemcpyDeviceToHost);
   hipMemcpy(err,_Derr,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   // Free memory on Device 
   hipFree(_Dt);
   hipFree(_Dy);
   hipFree(_Djt);
   hipFree(_Dneq);
   hipFree(_Dliw);
   hipFree(_Dlrw);
   hipFree(_Datol);
   hipFree(_Ditol);
   hipFree(_Diopt);
   hipFree(_Drtol);
   hipFree(_Dtout);
   hipFree(_Ditask);
   hipFree(_Diwork);
   hipFree(_Drwork);
   hipFree(_Distate);
   hipFree(_Dcommon);
   hipFree(_Derr);
   free(t);
   free(y);
   free(jt);
   free(neq);
   free(liw);
   free(lrw);
   free(atol);
   free(itol);
   free(iopt);
   free(rtol);
   free(tout);
   free(itask);
   free(iwork);
   free(rwork);
   free(istate);
   //free(common);
   free(err);
   //for ( int i = 0; i < td_h.size(); i++) cout << " " << td_h[i] << "   " << xd_h[i] << endl;
   if ( check_int )
   {
        //if ( (error_sim_h[j] < 0.01) && (check == false) )
        if (error_sim_h[0] < 0.003) 
        {  
           for ( int i = 0; i < td_h.size(); i++) cout << " " << td_h[i] << "   " << xd_h[i] << endl;
        }
   }
}
// Functor used to update parameters
template < class Tuple > 
void update_functor::operator()( Tuple step )
{
   temp_p = thrust::get<0>(step);
   temp_error = thrust::get<1>(step);
   temp_err = thrust::get<2>(step);
   //if (m_t_rand < temp_p)
   //if ( (m_t_rand < thrust::get<0>(step)) && (thrust::get<1>(step) > thrust::get<2>(step)) )    
   //if ( (m_t_rand < temp_p) && (temp_error > temp_err) )    
   if (temp_error > temp_err)    
   {
       //update error, r0, d, ea
       thrust::get<1>(step) = temp_err;
       thrust::get<3>(step) = thrust::get<4>(step);
       thrust::get<5>(step) = thrust::get<6>(step);
       thrust::get<7>(step) = thrust::get<8>(step);
       thrust::get<9>(step) = 1;
   }
} 
//MC Simulation
//void MC_sim ( const vector <double> &x_d, const vector <double> &t_d, const vector <double> &sd_d, const int *n_ka,  myFex &fex, const myJex &jex, state_type &error_opt )
void MC_sim ( const int &gene_ind, const vector <double> &x_d, const vector <double> &t_d, const double &mean_xd, const int *n_ka, const int *n_kd, const int &size_ka, const int &size_kd,  myFex &fex, const myJex &jex, state_type &error_opt )
{
   const int N_steps = 100;
   state_type r0_opt(probSize), d_opt(probSize), ea_opt(probSize), kaval_opt(size_ka), kdval_opt(size_kd);
   thrust::device_ptr<double> _Dr0_ptr, _Dd_ptr, _Dea_ptr, _Dkaval_ptr, _Dkdval_ptr, d_pt;
   thrust::counting_iterator<int> sequence_begin(0);
   fex.get_r0_ptr(_Dr0_ptr);
   thrust::copy(_Dr0_ptr, _Dr0_ptr+probSize, r0_opt.begin()); 
   fex.get_d_ptr(_Dd_ptr);
   thrust::copy(_Dd_ptr, _Dd_ptr+probSize, d_opt.begin());
   fex.get_ea_ptr(_Dea_ptr);
   thrust::copy(_Dea_ptr, _Dea_ptr+probSize, ea_opt.begin()); 
   fex.get_kaval_ptr(_Dkaval_ptr);
   thrust::copy(_Dkaval_ptr, _Dkaval_ptr+size_ka, kaval_opt.begin()); 
   fex.get_kdval_ptr(_Dkdval_ptr);
   thrust::copy(_Dkdval_ptr, _Dkdval_ptr+size_kd, kdval_opt.begin()); 
   state_type error_sim(probSize);
   srand(time(NULL));
   for ( int k =0; k < N_steps; k++)
   {
      // update param values
      thrust::transform(sequence_begin,sequence_begin + probSize,r0_opt.begin(),_Dr0_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + probSize,d_opt.begin(),_Dd_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + probSize,ea_opt.begin(),_Dea_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + size_ka,kaval_opt.begin(),_Dkaval_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + size_kd,kdval_opt.begin(),_Dkdval_ptr,move_functor());
      //fex.set_r0_ptr (_Dr0_ptr); 
      //fex.set_d_ptr (_Dd_ptr); 
      //fex.set_ea_ptr (_Dea_ptr); 
      //fex.set_ka_ptr (_Dkaval_ptr,size_ka); 
      //fex.set_kd_ptr (_Dkdval_ptr,size_kd); 
      integrate_lsoda_ode (gene_ind, x_d, t_d, mean_xd, fex, jex, error_sim);
      // Simulated Annealing calculation
      state_type p_MC(probSize); 
      double S_MC = 0.1 * exp( -2.0 * double(k+1)/double(N_steps));
      thrust::transform(error_opt.begin(),error_opt.end(),error_sim.begin(),p_MC.begin(),p_functor(S_MC));
      double temp_rand = abs(double(rand()%100)/100.0);
      // update_track 0 = no update, 1 = update
      thrust::device_vector<int> update_track(probSize);
      thrust::fill( update_track.begin(), update_track.end(), 0 );
      thrust::for_each( thrust::make_zip_iterator(
      thrust::make_tuple(
      p_MC.begin(),error_opt.begin(),error_sim.begin(),
      r0_opt.begin(),_Dr0_ptr,ea_opt.begin(),_Dea_ptr,d_opt.begin(),_Dd_ptr,update_track.begin())), 
      thrust::make_zip_iterator(
      thrust::make_tuple(
      p_MC.end(),error_opt.end(),error_sim.end(),
      r0_opt.end(),_Dr0_ptr+probSize,ea_opt.end(),_Dea_ptr+probSize,d_opt.end(),_Dd_ptr+probSize,update_track.end())),
              update_functor(temp_rand) );
      // update kavals now
      int sum_check = thrust::reduce(update_track.begin(),update_track.end());
      int ka_index = 0;
      int kd_index = 0;
      //host_type error_sim_h;
      //thrust::host_vector <int> update_track_h;
      //update_track_h =  update_track;
      //error_sim_h = error_sim;
      if ( sum_check > 0)
      {
         for (int k = 0; k < probSize; k++)
         {
             if ( update_track[k] == 1 )
             {
                //if (error_sim_h[k] < 0.1)cout << "Update for thread " << k << " with error of: " << error_sim_h[k] << endl;
                for (int j = 0; j < n_ka[k]; j++ )
                {
                    kaval_opt[ka_index] = _Dkaval_ptr[ka_index];
                    ka_index++;
                }
                for (int j = 0; j < n_kd[k]; j++ )
                {
                    kdval_opt[kd_index] = _Dkdval_ptr[kd_index];
                    kd_index++;
                }
             }  
             else 
             { 
                ka_index += n_ka[k];
                kd_index += n_kd[k];
             }
         }
      }
   }
   // update param values to optimized parameters in class fex
   thrust::copy(r0_opt.begin(),r0_opt.end(),_Dr0_ptr); 
   thrust::copy(d_opt.begin(),d_opt.end(),_Dd_ptr); 
   thrust::copy(ea_opt.begin(),ea_opt.end(),_Dea_ptr); 
   thrust::copy(kaval_opt.begin(),kaval_opt.end(),_Dkaval_ptr); 
   thrust::copy(kdval_opt.begin(),kdval_opt.end(),_Dkdval_ptr); 
}
