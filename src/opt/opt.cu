#include "hip/hip_runtime.h"
# include "../common.h"
using namespace std;
# include "../lsoda/cuLsoda_all.cu"
# include "../lsoda/cuLsoda.hpp"
# include "./opt.hpp"
// A guassian distribution with mean 0 standard deviation of 1
double guassrand()
{
      static double V1, V2, S;
      static int phase = 0;
      double X;

      srand(time(NULL));
      if (phase == 0) 
      {
      	 do {
	    double U1 = (double)rand() / RAND_MAX;
	    double U2 = (double)rand() / RAND_MAX;
            
            V1 = 2 * U1 - 1;
	    V2 = 2 * U2 - 1;
	    S = V1 * V1 + V2 * V2;
	    } while(S >= 1 || S == 0);

	    X = V1 * sqrt(-2 * log(S) / S);
      } else
      X = V2 * sqrt(-2 * log(S) / S);
      
      phase = 1 - phase;

      return X;
}
// LSODE integrator function
//void integrate_lsoda_ode ( const vector <double> &x_d, const vector <double> &t_d, const vector <double> &sd_d, const myFex &fex, const myJex &jex, state_type &error_sim_d)
void integrate_lsoda_ode ( const int &gene_ind, const vector <double> &x_d, const vector <double> &t_d, const double &mean_xd, const myFex &fex, const myJex &jex, state_type &error_sim_d)
{
   bool check_int = false; 
   host_type error_sim_h(probSize);
   //state_type error_sim_d(probSize);
   // Studying gene 20 for the N= 25 set
   /* Local variables: input arguments for Lsoda.
      For a more detailed description see cuLsoda.cu L 171*/
   // initial value of independent variable t
   double *t = (double*)malloc(sizeof(double)*probSize);
   // initial value of dependent variable Y.SIZE() = NEQ 
   double *y = (double*)malloc(sizeof(double)*probSize);
   // JT is the jacobian type indicator
   int *jt = (int*)malloc(sizeof(int)*probSize);
   // NEQ is the number of equations
   int *neq = (int*)malloc(sizeof(int)*probSize);
   // lengths of IWORK and RWORK
   int *liw = (int*)malloc(sizeof(int)*probSize);
   int *lrw = (int*)malloc(sizeof(int)*probSize);
   // ATOL is the absolute tolerance parameter
   double *atol = (double*)malloc(sizeof(double)*probSize);
   // ITOL size of ATOL, can be same or different for each EQ in NEQ
   int *itol =(int*) malloc(sizeof(int)*probSize);
   // IOPT optional inputs argument
   int *iopt =(int*) malloc(sizeof(int)*probSize);
   // RTOL relative tolerance parameter
   double *rtol = (double*)malloc(sizeof(double)*probSize);
   // IOUT forward step iterations   
   //int *iout =(int*) malloc(sizeof(int)*probSize);
   // TOUT time point where output is desired    
   double *tout =(double*) malloc(sizeof(double)*probSize);
   // ITASK determines normal computation of Y at TOUT
   int *itask = (int*)malloc(sizeof(int)*probSize);
   // IWORK int array of length of at least 20 + NEQ
   int *iwork =(int*) malloc(sizeof(int)*21*probSize);
   // RWORK array of length of at least 22 + NEQ * MAX(16, NEQ + 9)
   double *rwork = (double*)malloc(sizeof(double)*38*probSize);
   // ISTATE input flag  
   int *istate = (int*)malloc(sizeof(int)*probSize);
   // COMMON BLOCK DECLARATIONS
   struct cuLsodaCommonBlock common[probSize];
   struct cuLsodaCommonBlock *Hcommon = common;
   int *err = (int*)malloc(sizeof(int)*probSize);
   //  End Local Block 

   // Pointers to Device versions of Local variables 
   double *_Dt;
   double *_Dy;	// [3]
   int *_Djt;
   int *_Dneq;
   int *_Dliw;
   int *_Dlrw;
   double *_Datol;	//[3]
   int *_Ditol;
   int *_Diopt;
   double *_Drtol;
   double *_Dtout;
   int *_Ditask;
   int *_Diwork;	// [23]
   double *_Drwork;	// [70]
   int *_Distate;
   struct cuLsodaCommonBlock *_Dcommon;
   int *_Derr;
   // End Pointer Block 
   // Transfer other data to device
   state_type x_d_d = x_d;
   state_type t_d_d = t_d;
   //state_type sd_d_d = sd_d;
   host_type xd_h, td_h;
   //host_type y_val_h(N_time_points*probSize);
	
   // Method instantiations for Derivative and Jacobian functions to send to template 
   thrust::device_ptr<double> t0_ptr;
   thrust::device_ptr<int> index_ptr;
   thrust::fill(error_sim_d.begin(), error_sim_d.end(), 0.0);
   // Assignment of initial values to locals 
   for (int i = 0; i < probSize; i++)
   {
      *(neq+i) = 1;
      *(y+0+i) = x_d[gene_ind*N_time_points];
      //*(y+0+i) = 1.0;
      *(t+i) = (double)0.;
      *(tout+i) = 1.0;
      *(itol+i) = 1;
      *(rtol+i) = (double)1e-3;
      *(atol+i) = (double)1e-6;
      *(itask+i) = 1;
      *(istate+i) = 1;
      *(iopt+i) = 0;
      *(lrw+i) = 38;
      *(liw+i) = 21;
      *(jt+i) = 2;
      cuLsodaCommonBlockInit(&Hcommon[i]);
      *(err +i) = -1;
   } 
   // Allocate device memory for each of the pointers, and copy the values from local to device
   hipMalloc((void**)&_Dt,sizeof(double)*probSize);
   hipMemcpy(_Dt,t,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dy,sizeof(double)*probSize);							
   hipMemcpy(_Dy,y,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Djt,sizeof(int)*probSize);
   hipMemcpy(_Djt,jt,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dneq,sizeof(int)*probSize);
   hipMemcpy(_Dneq,neq,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dliw,sizeof(int)*probSize);
   hipMemcpy(_Dliw,liw,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dlrw,sizeof(int)*probSize);
   hipMemcpy(_Dlrw,lrw,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Datol,sizeof(double)*probSize);
   hipMemcpy(_Datol,atol,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Ditol,sizeof(int)*probSize);							
   hipMemcpy(_Ditol,itol,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Diopt,sizeof(int)*probSize);							
   hipMemcpy(_Diopt,iopt,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Drtol,sizeof(double)*probSize);							
   hipMemcpy(_Drtol,rtol,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dtout,sizeof(double)*probSize);
   hipMemcpy(_Dtout,tout,sizeof(double)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Ditask,sizeof(int)*probSize);
   hipMemcpy(_Ditask,itask,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Diwork,sizeof(int)*21*probSize);
   hipMemcpy(_Diwork,iwork,sizeof(int)*21*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Drwork,sizeof(double)*38*probSize);
   hipMemcpy(_Drwork,rwork,sizeof(double)*38*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Distate,sizeof(int)*probSize);							
   hipMemcpy(_Distate,istate,sizeof(int)*probSize,hipMemcpyHostToDevice);
   hipMalloc((void**)&_Dcommon,sizeof(struct cuLsodaCommonBlock)*probSize);	
   hipMemcpy(_Dcommon,Hcommon,sizeof(struct cuLsodaCommonBlock)*probSize, hipMemcpyHostToDevice);
   hipMalloc((void**)&_Derr,sizeof(double)*probSize);
   hipMemcpy(_Derr,istate,sizeof(double)*probSize,hipMemcpyHostToDevice);
   // End Allocation and Copy Block 
   thrust::device_ptr<double> y_dev_p = thrust::device_pointer_cast(_Dy);
   thrust::device_ptr<double> t_dev_p = thrust::device_pointer_cast(_Dt);
   thrust::device_ptr<double> tout_dev_p = thrust::device_pointer_cast(_Dtout);
   cuLsoda<<<blocksPerGrid,threadsPerBlock>>>(fex, _Dneq, _Dy, _Dt, _Dtout, _Ditol, _Drtol, _Datol, _Ditask, _Distate, _Diopt, _Drwork, _Dlrw, _Diwork, _Dliw, jex, _Djt, _Dcommon, _Derr, probSize);
   //thrust::copy(y_dev_p,y_dev_p+probSize,y_val_h.begin());
   //xd_h.push_back(y_dev_p[0]);
   //td_h.push_back(tout_dev_p[0]);
   thrust::fill(tout_dev_p, tout_dev_p+probSize, t_d_d[1]);
   thrust::fill(y_dev_p, y_dev_p+probSize, x_d_d[gene_ind*N_time_points]);
   //thrust::fill(y_dev_p, y_dev_p+probSize, 1.0);
   thrust::fill(t_dev_p, t_dev_p+probSize, 0.0);
   //for ( int t_ind = 1; t_ind <= N_time_points; t_ind++)
   int t_ind = 1; 
   while ( t_ind < N_time_points )
   {
       cuLsoda<<<blocksPerGrid,threadsPerBlock>>>(fex, _Dneq, _Dy, _Dt, _Dtout, _Ditol, _Drtol, _Datol, _Ditask, _Distate, _Diopt, _Drwork, _Dlrw, _Diwork, _Dliw, jex, _Djt, _Dcommon, _Derr, probSize);
       if (check_int)
       {
          xd_h.push_back(y_dev_p[0]);
          td_h.push_back(tout_dev_p[0]);
       }
       thrust::fill(tout_dev_p, tout_dev_p+probSize, t_d_d[t_ind+1]);
       thrust::transform(error_sim_d.begin(), error_sim_d.end(), y_dev_p, 
                         error_sim_d.begin(), error_functor(x_d_d[gene_ind*N_time_points+t_ind],mean_xd));
       //thrust::fill(y_dev_p, y_dev_p+probSize, 1.0);
       //thrust::fill(t_dev_p, t_dev_p+probSize, 0.0);
       t_ind++;
   }
   error_sim_h = error_sim_d;
   // Copy memory back from Device to Host 
   hipMemcpy(t,_Dt,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(y,_Dy,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(jt,_Djt,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(neq,_Dneq,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(liw,_Dliw,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(lrw,_Dlrw,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(atol,_Datol,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(itol,_Ditol,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(iopt,_Diopt,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(rtol,_Drtol,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(tout,_Dtout,sizeof(double)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(itask,_Ditask,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(iwork,_Diwork,sizeof(int)*21*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(rwork,_Drwork,sizeof(double)*38*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(istate,_Distate,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   hipMemcpy(Hcommon,_Dcommon,sizeof(struct cuLsodaCommonBlock)*probSize, hipMemcpyDeviceToHost);
   hipMemcpy(err,_Derr,sizeof(int)*probSize,hipMemcpyDeviceToHost);
   // Free memory on Device 
   hipFree(_Dt);
   hipFree(_Dy);
   hipFree(_Djt);
   hipFree(_Dneq);
   hipFree(_Dliw);
   hipFree(_Dlrw);
   hipFree(_Datol);
   hipFree(_Ditol);
   hipFree(_Diopt);
   hipFree(_Drtol);
   hipFree(_Dtout);
   hipFree(_Ditask);
   hipFree(_Diwork);
   hipFree(_Drwork);
   hipFree(_Distate);
   hipFree(_Dcommon);
   hipFree(_Derr);
   free(t);
   free(y);
   free(jt);
   free(neq);
   free(liw);
   free(lrw);
   free(atol);
   free(itol);
   free(iopt);
   free(rtol);
   free(tout);
   free(itask);
   free(iwork);
   free(rwork);
   free(istate);
   //free(common);
   free(err);
   //for ( int i = 0; i < td_h.size(); i++) cout << " " << td_h[i] << "   " << xd_h[i] << endl;
   if ( check_int )
   {
        //if ( (error_sim_h[j] < 0.01) && (check == false) )
        if (error_sim_h[0] < 0.003) 
        {  
           for ( int i = 0; i < td_h.size(); i++) cout << " " << td_h[i] << "   " << xd_h[i] << endl;
        }
   }
}
// Functor used to update parameters
template < class Tuple > 
void update_functor::operator()( Tuple step )
{
   temp_p = thrust::get<0>(step);
   temp_error = thrust::get<1>(step);
   temp_err = thrust::get<2>(step);
   //if (m_t_rand < temp_p)
   //if ( (m_t_rand < thrust::get<0>(step)) && (thrust::get<1>(step) > thrust::get<2>(step)) )    
   //if ( (m_t_rand < temp_p) && (temp_error > temp_err) )    
   if (temp_error > temp_err)    
   {
       //update error, r0, d, ea
       thrust::get<1>(step) = temp_err;
       thrust::get<3>(step) = thrust::get<4>(step);
       thrust::get<5>(step) = thrust::get<6>(step);
       thrust::get<7>(step) = thrust::get<8>(step);
       thrust::get<9>(step) = 1;
   }
} 
//MC Simulation
//void MC_sim ( const vector <double> &x_d, const vector <double> &t_d, const vector <double> &sd_d, const int *n_ka,  myFex &fex, const myJex &jex, state_type &error_opt )
void MC_sim ( const int &gene_ind, const vector <double> &x_d, const vector <double> &t_d, const double &mean_xd, const int *n_ka, const int *n_kd, const int &size_ka, const int &size_kd,  myFex &fex, const myJex &jex, state_type &error_opt )
{
   const int N_steps = 100;
   state_type r0_opt(probSize), d_opt(probSize), ea_opt(probSize), kaval_opt(size_ka), kdval_opt(size_kd);
   thrust::device_ptr<double> _Dr0_ptr, _Dd_ptr, _Dea_ptr, _Dkaval_ptr, _Dkdval_ptr, d_pt;
   thrust::counting_iterator<int> sequence_begin(0);
   fex.get_r0_ptr(_Dr0_ptr);
   thrust::copy(_Dr0_ptr, _Dr0_ptr+probSize, r0_opt.begin()); 
   fex.get_d_ptr(_Dd_ptr);
   thrust::copy(_Dd_ptr, _Dd_ptr+probSize, d_opt.begin());
   fex.get_ea_ptr(_Dea_ptr);
   thrust::copy(_Dea_ptr, _Dea_ptr+probSize, ea_opt.begin()); 
   fex.get_kaval_ptr(_Dkaval_ptr);
   thrust::copy(_Dkaval_ptr, _Dkaval_ptr+size_ka, kaval_opt.begin()); 
   fex.get_kdval_ptr(_Dkdval_ptr);
   thrust::copy(_Dkdval_ptr, _Dkdval_ptr+size_kd, kdval_opt.begin()); 
   state_type error_sim(probSize);
   srand(time(NULL));
   for ( int k =0; k < N_steps; k++)
   {
      // update param values
      thrust::transform(sequence_begin,sequence_begin + probSize,r0_opt.begin(),_Dr0_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + probSize,d_opt.begin(),_Dd_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + probSize,ea_opt.begin(),_Dea_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + size_ka,kaval_opt.begin(),_Dkaval_ptr,move_functor());
      thrust::transform(sequence_begin,sequence_begin + size_kd,kdval_opt.begin(),_Dkdval_ptr,move_functor());
      //fex.set_r0_ptr (_Dr0_ptr); 
      //fex.set_d_ptr (_Dd_ptr); 
      //fex.set_ea_ptr (_Dea_ptr); 
      //fex.set_ka_ptr (_Dkaval_ptr,size_ka); 
      //fex.set_kd_ptr (_Dkdval_ptr,size_kd); 
      integrate_lsoda_ode (gene_ind, x_d, t_d, mean_xd, fex, jex, error_sim);
      // Simulated Annealing calculation
      state_type p_MC(probSize); 
      double S_MC = 0.1 * exp( -2.0 * double(k+1)/double(N_steps));
      thrust::transform(error_opt.begin(),error_opt.end(),error_sim.begin(),p_MC.begin(),p_functor(S_MC));
      double temp_rand = abs(double(rand()%100)/100.0);
      // update_track 0 = no update, 1 = update
      thrust::device_vector<int> update_track(probSize);
      thrust::fill( update_track.begin(), update_track.end(), 0 );
      thrust::for_each( thrust::make_zip_iterator(
      thrust::make_tuple(
      p_MC.begin(),error_opt.begin(),error_sim.begin(),
      r0_opt.begin(),_Dr0_ptr,ea_opt.begin(),_Dea_ptr,d_opt.begin(),_Dd_ptr,update_track.begin())), 
      thrust::make_zip_iterator(
      thrust::make_tuple(
      p_MC.end(),error_opt.end(),error_sim.end(),
      r0_opt.end(),_Dr0_ptr+probSize,ea_opt.end(),_Dea_ptr+probSize,d_opt.end(),_Dd_ptr+probSize,update_track.end())),
              update_functor(temp_rand) );
      // update kavals now
      int sum_check = thrust::reduce(update_track.begin(),update_track.end());
      int ka_index = 0;
      int kd_index = 0;
      //host_type error_sim_h;
      //thrust::host_vector <int> update_track_h;
      //update_track_h =  update_track;
      //error_sim_h = error_sim;
      if ( sum_check > 0)
      {
         for (int k = 0; k < probSize; k++)
         {
             if ( update_track[k] == 1 )
             {
                //if (error_sim_h[k] < 0.1)cout << "Update for thread " << k << " with error of: " << error_sim_h[k] << endl;
                for (int j = 0; j < n_ka[k]; j++ )
                {
                    kaval_opt[ka_index] = _Dkaval_ptr[ka_index];
                    ka_index++;
                }
                for (int j = 0; j < n_kd[k]; j++ )
                {
                    kdval_opt[kd_index] = _Dkdval_ptr[kd_index];
                    kd_index++;
                }
             }  
             else 
             { 
                ka_index += n_ka[k];
                kd_index += n_kd[k];
             }
         }
      }
   }
   // update param values to optimized parameters in class fex
   thrust::copy(r0_opt.begin(),r0_opt.end(),_Dr0_ptr); 
   thrust::copy(d_opt.begin(),d_opt.end(),_Dd_ptr); 
   thrust::copy(ea_opt.begin(),ea_opt.end(),_Dea_ptr); 
   thrust::copy(kaval_opt.begin(),kaval_opt.end(),_Dkaval_ptr); 
   thrust::copy(kdval_opt.begin(),kdval_opt.end(),_Dkdval_ptr); 
}/*
//Nelder-Mead Optimization
void NM_sim( const state_type &x_d, const state_type &t_d, const state_type &mean_xd, myFex_single &fex_nm, myJex_single &jex_nm, const int &gene_ind, const int nka, const int nkd, double &error_ode, state_type &param)
{
    const int param_size = 3 + nka + nkd;
    // Local optimization search in log10 space
    //transform(param.begin(),param.end(),param.begin(),logb10());
    // Nelder Mead optimization
    double reqmin = 1.0E-08;
    vector <double> step(param_size,1.0);
    int konvge = 10;
    int kcount = 100;
    double ccoeff = 0.5;
    double del;
    double dn;
    double dnn;
    double ecoeff = 2.0;
    double eps = 0.001;
    int ihi;
    int ilo;
    int jcount;
    int l_min;
    int nn;
    int ifault_nm, icount_nm, numres_nm;
    double ynewlo;
    vector <double> xmin(param_size);
    double rcoeff = 1.0;
    double rq;
    double x_min;
    vector <double> y_min(param_size+1);
    double y2star;
    double ylo;
    double ystar;
    double z_min;
    double err = 0.0;
//  Check the input parameters.
    if ( reqmin <= 0.0 )
    {
      ifault_nm = 1;
     // return;
    }
    if ( param_size < 1 )
    {
      ifault_nm = 1;
      //return;
    }
    if ( konvge < 1 )
    {
      ifault_nm = 1;
      //return;
    }
    //
    vector <value_type> p_min(param_size*(param_size+1));
    vector <value_type> p2star(param_size);
    vector <value_type> pbar(param_size);
    vector <value_type> pstar(param_size);
    icount_nm = 0;
    numres_nm = 0;
    jcount = konvge; 
    dn = ( double ) ( param_size );
    nn = param_size + 1;
    dnn = ( double ) ( nn );
    del = 0.01;
    rq = reqmin * dn;
//  Initial or restarted loop.
    for ( ; ; )
    {
      for ( int i = 0; i < param_size; i++ )
      { 
        p_min[i+param_size*param_size] = param[i];
      }
      y_min[param_size] = error_ode;

      for ( int j = 0; j < param_size; j++ )
      {
        x_min = param[j];
        param[j] += step[j] * del;
        for ( int i = 0; i < param_size; i++ )
        {
          p_min[i+j*param_size] = param[i];
        }
        integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, param );
        y_min[j] = err;
        icount_nm += 1;
        param[j] = x_min;
      }
////  The simplex construction is complete.
////  Find highest and lowest Y values.  YNEWLO = Y(IHI) indicates
////  the vertex of the simplex to be replaced.
      ylo = y_min[0];
      ilo = 0;

      for ( int i = 1; i < nn; i++ )
      {
        if ( y_min[i] < ylo )
        {
          ylo = y_min[i];
          ilo = i;
        }
      }
////  Inner loop.
      for ( ; ; )
      {
        if ( kcount <= icount_nm )
        {
          break;
        }
        ynewlo = y_min[0];
        ihi = 0;

        for ( int i = 1; i < nn; i++ )
        {
          if ( ynewlo < y_min[i] )
          {
            ynewlo = y_min[i];
            ihi = i;
          }
        }
////  Calculate PBAR, the centroid of the simplex vertices
////  excepting the vertex with Y value YNEWLO.
        for ( int i = 0; i < param_size; i++ )
        {
          z_min = 0.0;
          for ( int j = 0; j < nn; j++ )
          { 
            z_min = z_min + p_min[i+j*param_size];
          }
          z_min = z_min - p_min[i+ihi*param_size];  
          pbar[i] = z_min / dn;
        }
////  Reflection through the centroid.
        for ( int i = 0; i < param_size; i++ )
        {
          pstar[i] = pbar[i] + rcoeff * ( pbar[i] - p_min[i+ihi*param_size] );
        }
        // Calculate error with new parameters
        integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, pstar );
        ystar = err;
        icount_nm += 1;
////  Successful reflection, so extension.
        if ( ystar < ylo )
        {
          for ( int i = 0; i < param_size; i++ )
          {
            p2star[i] = pbar[i] + ecoeff * ( pstar[i] - pbar[i] );
          }
          // Calculate error with new parameters
          integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, p2star );
          y2star = err;
          icount_nm += 1;
////  Check extension.
          if ( ystar < y2star )
          {
            for ( int i = 0; i < param_size; i++ )
            {
              p_min[i+ihi*param_size] = pstar[i];
            }
            y_min[ihi] = ystar;
          }
////  Retain extension or contraction.
          else
          {
            for ( int i = 0; i < param_size; i++ )
            {
              p_min[i+ihi*param_size] = p2star[i];
            }
            y_min[ihi] = y2star;
          }
        }
////  No extension.
        else
        {
          l_min = 0;
          for ( int i = 0; i < nn; i++ )
          {
            if ( ystar < y_min[i] )
            {
              l_min += 1;
            }
          }

          if ( 1 < l_min )
          {
            for ( int i = 0; i < param_size; i++ )
            {
              p_min[i+ihi*param_size] = pstar[i];
            }
            y_min[ihi] = ystar;
          }
////  Contraction on the Y(IHI) side of the centroid.
          else if ( l_min == 0 )
          {
            for ( int i = 0; i < param_size; i++ )
            {
              p2star[i] = pbar[i] + ccoeff * ( p_min[i+ihi*param_size] - pbar[i] );
            }
            integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, p2star );
            y2star = err;
            icount_nm += 1;
////  Contract the whole simplex.
            if ( y_min[ihi] < y2star )
            {
              for ( int j = 0; j < nn; j++ )
              {
                for ( int i = 0; i < param_size; i++ )
                {
                  p_min[i+j*param_size] = ( p_min[i+j*param_size] + p_min[i+ilo*param_size] ) * 0.5;
                  xmin[i] = p_min[i+j*param_size];
                }
                 integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, xmin );
                 y2star = err;
                 icount_nm += 1;
              }
              ylo = y_min[0];
              ilo = 0;

              for ( int i = 1; i < nn; i++ )
              {
                if ( y_min[i] < ylo )
                {
                  ylo = y_min[i];
                  ilo = i;
                }
              }
              continue;
            }
////  Retain contraction.
            else
            {
              for ( int i = 0; i < param_size; i++ )
              {
                p_min[i+ihi*param_size] = p2star[i];
              }
              y_min[ihi] = y2star;
            }
          }
////  Contraction on the reflection side of the centroid.
          else if ( l_min == 1 )
          {
            for ( int i = 0; i < param_size; i++ )
            {
              p2star[i] = pbar[i] + ccoeff * ( pstar[i] - pbar[i] );
            }
            integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, p2star );
            y2star = err;
            icount_nm += 1;
////  Retain reflection?
            if ( y2star <= ystar )
            {
              for ( int i = 0; i < param_size ; i++ )
              {
                p_min[i+ihi*param_size] = p2star[i];
              }
              y_min[ihi] = y2star;
            }
            else
            {
              for ( int i = 0; i < param_size; i++ )
              {
                p_min[i+ihi*param_size] = pstar[i];
              }
              y_min[ihi] = ystar;
            }
          }
        }
////  Check if YLO improved.
        if ( y_min[ihi] < ylo )
        {
          ylo = y_min[ihi];
          ilo = ihi;
        }
        jcount -= 1;

        if ( 0 < jcount )
        {
          continue;
        }
////  Check to see if minimum reached.
        if ( icount_nm <= kcount )
        {
          jcount = konvge;

          z_min = 0.0;
          for ( int i = 0; i < nn; i++ )
          {
            z_min = z_min + y_min[i];
          }
          x_min = z_min / dnn;

          z_min = 0.0;
          for ( int i = 0; i < nn; i++ )
          {
            z_min = z_min + pow ( y_min[i] - x_min, 2 );
          }

          if ( z_min <= rq )
          {
            break;
          }
        }
      }
////  Factorial tests to check that YNEWLO is a local minimum.
      for ( int i = 0; i < param_size; i++ )
      {
        xmin[i] = p_min[i+ilo*param_size];
      }
      ynewlo = y_min[ilo];

      if ( kcount < icount_nm )
      {
        ifault_nm = 2;
        break;
      }

      ifault_nm = 0;

      for ( int i = 0; i < param_size; i++ )
      {
        del = step[i] * eps;
        xmin[i] = xmin[i] + del;
        integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, xmin );
        z_min = err;
        icount_nm += 1;
        if ( z_min < ynewlo )
        {
          ifault_nm = 2;
          break;
        }
        xmin[i] = xmin[i] - del - del;
        integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, xmin );
        z_min = err;
        icount_nm += 1;
        if ( z_min < ynewlo )
        {
          ifault_nm = 2;
          break;
        }
        xmin[i] = xmin[i] + del;
      }

      if ( ifault_nm == 0 )
      {
        break;
      }
////  Restart the procedure.
      for ( int i = 0; i < param_size; i++ )
      {
        param[i] = xmin[i];
      }
      del = eps;
      numres_nm += 1;
    }
    //integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, xmin );
    integrate_lsoda_nm ( x_d, t_d, mean_xd, fex_nm, jex_nm, gene_ind, nka, nkd, err, param );
    //cout << "The optimized value of error after NM is: " << err << endl;
    error_ode = err;
    //cout << "-----------NM_Sim_end------------- " << endl;
    param = xmin;
};*/
