#include "hip/hip_runtime.h"
#include "../common.h"
using namespace std;
# include "./spline.hpp"
//****************************************************************************80

double pchst ( double arg1, double arg2 )

//****************************************************************************80
//
//  Purpose:
//
//    PCHST: PCHIP sign-testing routine.
//
//  Discussion:
//
//    This routine essentially computes the sign of ARG1 * ARG2.
//
//    The object is to do this without multiplying ARG1 * ARG2, to avoid
//    possible over/underflow problems.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license. 
//
//  Modified:
//
//    12 August 2005
//
//  Author:
//
//    Original FORTRAN77 version by Fred Fritsch, Lawrence Livermore National Laboratory.
//    C++ version by John Burkardt.
//
//  Reference:
//
//    Fred Fritsch, Ralph Carlson, 
//    Monotone Piecewise Cubic Interpolation,
//    SIAM Journal on Numerical Analysis,
//    Volume 17, Number 2, April 1980, pages 238-246.
//
//  Parameters:
//
//    Input, double ARG1, ARG2, two values to check.
//
//    Output, double PCHST,
//    -1.0, if ARG1 and ARG2 are of opposite sign.
//     0.0, if either argument is zero.
//    +1.0, if ARG1 and ARG2 are of the same sign.
//
{
  double value;

  if ( arg1 == 0.0 )
  {
    value = 0.0;
  }
  else if ( arg1 < 0.0 )
  {
    if ( arg2 < 0.0 )
    {
      value = 1.0;
    }
    else if ( arg2 == 0.0 )
    {
      value = 0.0;
    }
    else if ( 0.0 < arg2 )
    {
      value = -1.0;
    }
  }
  else if ( 0.0 < arg1 )
  {
    if ( arg2 < 0.0 )
    {
      value = -1.0;
    }
    else if ( arg2 == 0.0 )
    {
      value = 0.0;
    }
    else if ( 0.0 < arg2 )
    {
      value = 1.0;
    }
  }

  return value;
}
//****************************************************************************80

double r8_max ( double x, double y )

//****************************************************************************80
//
//  Purpose:
//
//    R8_MAX returns the maximum of two R8's.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license. 
//
//  Modified:
//
//    10 January 2002
//
//  Author:
//
//    John Burkardt
//
//  Parameters:
//
//    Input, double X, Y, the quantities to compare.
//
//    Output, double R8_MAX, the maximum of X and Y.
//
{
  if ( y < x )
  {
    return x;
  } 
  else
  {
    return y;
  }
}
//****************************************************************************80

double r8_min ( double x, double y )

//****************************************************************************80
//
//  Purpose:
//
//    R8_MIN returns the minimum of two R8's.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license. 
//
//  Modified:
//
//    09 May 2003
//
//  Author:
//
//    John Burkardt
//
//  Parameters:
//
//    Input, double X, Y, the quantities to compare.
//
//    Output, double R8_MIN, the minimum of X and Y.
//
{
  if ( y < x )
  {
    return y;
  } 
  else
  {
    return x;
  }
}
//****************************************************************************80

void spline_pchip_set ( const int n, const vector <double> &x, const vector <double> &f, vector <double> &d )

//****************************************************************************80
//
//  Purpose:
//
//    SPLINE_PCHIP_SET sets derivatives for a piecewise cubic Hermite interpolant.
//
//  Discussion:
//
//    This routine computes what would normally be called a Hermite 
//    interpolant.  However, the user is only required to supply function
//    values, not derivative values as well.  This routine computes
//    "suitable" derivative values, so that the resulting Hermite interpolant
//    has desirable shape and monotonicity properties.
//
//    The interpolant will have an extremum at each point where
//    monotonicity switches direction.
//
//    The resulting piecewise cubic Hermite function may be evaluated
//    by SPLINE_PCHIP_VAL..
//
//    This routine was originally called "PCHIM".
//
//    An "abs" was corrected to a "fabs" on the report of Thomas Beutlich,
//    10 October 2012.
//
//  Licensing:
//
//    This code is distributed under the GNU LGPL license. 
//
//  Modified:
//
//    14 August 2005
//
//  Author:
//
//    FORTRAN77 original version by Fred Fritsch, Lawrence Livermore National Laboratory.
//    C++ version by John Burkardt.
//
//  Reference:
//
//    Fred Fritsch, Ralph Carlson,
//    Monotone Piecewise Cubic Interpolation,
//    SIAM Journal on Numerical Analysis,
//    Volume 17, Number 2, April 1980, pages 238-246.
//
//    Fred Fritsch, Judy Butland,
//    A Method for Constructing Local Monotone Piecewise 
//    Cubic Interpolants,
//    SIAM Journal on Scientific and Statistical Computing,
//    Volume 5, Number 2, 1984, pages 300-304.
//
//  Parameters:
//
//    Input, int N, the number of data points.  N must be at least 2.
//
//    Input, double X[N], the strictly increasing independent
//    variable values.
//
//    Input, double F[N], dependent variable values to be interpolated.  This 
//    routine is designed for monotonic data, but it will work for any F-array.
//    It will force extrema at points where monotonicity switches direction.
//
//    Output, double D[N], the derivative values at the
//    data points.  If the data are monotonic, these values will determine
//    a monotone cubic Hermite function.  
//
{
  double del1;
  double del2;
  double dmax;
  double dmin;
  double drat1;
  double drat2;
  double dsave;
  double h1;
  double h2;
  double hsum;
  double hsumt3;
  int i;
  int ierr;
  int nless1;
  double temp;
  double w1;
  double w2;
//
//  Check the arguments.
//
  if ( n < 2 )
  {
    ierr = -1;
    cerr << "\n";
    cerr << "SPLINE_PCHIP_SET - Fatal error!\n";
    cerr << "  Number of data points less than 2.\n";
    exit ( ierr );
  }

  for ( i = 1; i < n; i++ )
  {
    if ( x[i] <= x[i-1] )
    {
      ierr = -3;
      cerr << "\n";
      cerr << "SPLINE_PCHIP_SET - Fatal error!\n";
      cerr << "  X array not strictly increasing.\n";
      exit ( ierr );
    }
  }

  ierr = 0;
  nless1 = n - 1;
  h1 = x[1] - x[0];
  del1 = ( f[1] - f[0] ) / h1;
  dsave = del1;
//
//  Special case N=2, use linear interpolation.
//
  if ( n == 2 )
  {
    d.push_back(del1);
    d.push_back(del1);
    return;
  }
//
//  Normal case, 3 <= N.
//
  h2 = x[2] - x[1];
  del2 = ( f[2] - f[1] ) / h2;
//
//  Set D(1) via non-centered three point formula, adjusted to be
//  shape preserving.
//
  hsum = h1 + h2;
  w1 = ( h1 + hsum ) / hsum;
  w2 = -h1 / hsum;
  d[0] = w1 * del1 + w2 * del2;

  if ( pchst ( d[0], del1 ) <= 0.0 )
  {
    d[0] = 0.0;
  }
//
//  Need do this check only if monotonicity switches.
//
  else if ( pchst ( del1, del2 ) < 0.0 )
  {
     dmax = 3.0 * del1;

     if ( fabs ( dmax ) < fabs ( d[0] ) )
     {
       d[0] = dmax;
     }

  }
//
//  Loop through interior points.
//
  for ( i = 2; i <= nless1; i++ )
  {
    if ( 2 < i )
    {
      h1 = h2;
      h2 = x[i] - x[i-1];
      hsum = h1 + h2;
      del1 = del2;
      del2 = ( f[i] - f[i-1] ) / h2;
    }
//
//  Set D(I)=0 unless data are strictly monotonic.
//
    d[i-1] = 0.0;

    temp = pchst ( del1, del2 );

    if ( temp < 0.0 )
    {
      ierr = ierr + 1;
      dsave = del2;
    }
//
//  Count number of changes in direction of monotonicity.
//
    else if ( temp == 0.0 )
    {
      if ( del2 != 0.0 )
      {
        if ( pchst ( dsave, del2 ) < 0.0 )
        {
          ierr = ierr + 1;
        }
        dsave = del2;
      }
    }
//
//  Use Brodlie modification of Butland formula.
//
    else
    {
      hsumt3 = 3.0 * hsum;
      w1 = ( hsum + h1 ) / hsumt3;
      w2 = ( hsum + h2 ) / hsumt3;
      dmax = r8_max ( fabs ( del1 ), fabs ( del2 ) );
      dmin = r8_min ( fabs ( del1 ), fabs ( del2 ) );
      drat1 = del1 / dmax;
      drat2 = del2 / dmax;
      d[i-1] = dmin / ( w1 * drat1 + w2 * drat2 );
    }
  }
//
//  Set D(N) via non-centered three point formula, adjusted to be
//  shape preserving.
//
  w1 = -h2 / hsum;
  w2 = ( h2 + hsum ) / hsum;
  d[n-1] = w1 * del1 + w2 * del2;

  if ( pchst ( d[n-1], del2 ) <= 0.0 )
  {
    d[n-1] = 0.0;
  }
  else if ( pchst ( del1, del2 ) < 0.0 )
  {
//
//  Need do this check only if monotonicity switches.
//
    dmax = 3.0 * del2;

    if ( fabs ( dmax ) < fabs ( d[n-1] ) )
    {
      d[n-1] = dmax;
    }

  }
  return;
}
//****************************************************************************80
void spline_deriv ( int gene_index, int N_time_s, vector <double> &t_d_s, vector <double> &x_d_s, vector <double> &deriv_s)
{
     const int start = gene_index*N_time_s;
     const int end = (gene_index + 1) * N_time_s;
     vector <double> x_d_temp ( x_d_s.begin () + start, x_d_s.begin () + end );
     //cout << "Size of x_d_temp: " << x_d_temp.size () << endl;
     vector <double> d_temp (N_time_s);
     spline_pchip_set ( N_time_s, t_d_s, x_d_temp, d_temp );
     //spline_pchip_set ( N_time_points, t_d, x_66, d_66 );
     for ( int i = 0; i < N_time_s; i++ )
     {
         deriv_s[start+i] = d_temp[i]; 
     }

     return;

}
//
void spline_coeff ( const int N_gene, const int N_time_s, const vector <double> &t_d, const vector <double> &x_d, const vector <double> &deriv_s, vector <double> &cub_coeff_spline)
{
   for ( int i = 0; i < N_gene; i++)
   {
       cub_coeff_spline[i*N_time_s*4] = x_d[i*N_time_s];
       // 
       for ( int j = 1; j < N_time_s ; j++ )
       {
           // Estimate the coefficient of the piecewise cubic spline
           double df = 0.0;
           double h = 0.0;
           double del1, del2;

           h = t_d[j] - t_d[j-1];
           df = (x_d[i*N_time_s+j] - x_d[i*N_time_s+j-1]) / h;
 
           // p(x) = c0 + (x-x_0)*(c1+(x-x_0)*(c2+(x-x_0)*c3))
           cub_coeff_spline[i*N_time_s*4+j*4+0] = x_d[i*N_time_s+j-1];
           cub_coeff_spline[i*N_time_s*4+j*4+1] = deriv_s[i*N_time_s+j-1];  
           del1 = (deriv_s[i*N_time_s+j-1] - df) / h;
           del2 = (deriv_s[i*N_time_s+j] - df) / h;
           cub_coeff_spline[i*N_time_s*4+j*4+2] = - ( del1 + del1 + del2);
           cub_coeff_spline[i*N_time_s*4+j*4+3] = (del1 + del2) / h;
        }
   }
}
//
void calc_spline( const vector <double> &x_d, const vector<double> &t_d, const vector<double> &cub_coeff, const int &ind, vector<double> &x_spline)
{
     x_spline.push_back(x_d[ind*N_time_points]);
     for (int t_ind = 1; t_ind < 14; t_ind++ )
     {
          double t_t = 0.0; 
          if ( (double(t_ind) > 0.0) && (double(t_ind) <= 1.1818) )
          {
             t_t = double(t_ind);
             x_spline.push_back(cub_coeff[ind*N_time_points*4+4]+t_t*
                        (cub_coeff[ind*N_time_points*4+4+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+4+2]+t_t*
                             cub_coeff[ind*N_time_points*4+4+3])));
          }
          else if ( (double(t_ind) > 1.1818) && (double(t_ind) <= 2.3636) )
          {
             t_t = double(t_ind)-1.1818;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+8]+t_t*
                        (cub_coeff[ind*N_time_points*4+8+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+8+2]+t_t*
                             cub_coeff[ind*N_time_points*4+8+3])));
          }
          else if ( (double(t_ind) > 2.3636) && (double(t_ind) <= 3.5455) )
          {
             t_t = double(t_ind)-2.3636;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+12]+t_t*
                        (cub_coeff[ind*N_time_points*4+12+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+12+2]+t_t*
                             cub_coeff[ind*N_time_points*4+12+3])));
          }
          else if ( (double(t_ind) > 3.5455) && (double(t_ind) <= 4.7273) )
          {
             t_t = double(t_ind)-3.5455;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+16]+t_t*
                        (cub_coeff[ind*N_time_points*4+16+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+16+2]+t_t*
                             cub_coeff[ind*N_time_points*4+16+3])));
          }
          else if ( (double(t_ind) > 4.7273) && (double(t_ind) <= 5.9091) )
          {
             t_t = double(t_ind)-4.7273;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+20]+t_t*
                        (cub_coeff[ind*N_time_points*4+20+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+20+2]+t_t*
                             cub_coeff[ind*N_time_points*4+20+3])));
          }
          else if ( (double(t_ind) > 5.9091) && (double(t_ind) <= 7.0909) )
          {
             t_t = double(t_ind)-5.9091;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+24]+t_t*
                        (cub_coeff[ind*N_time_points*4+24+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+24+2]+t_t*
                             cub_coeff[ind*N_time_points*4+24+3])));
          }
          else if ( (double(t_ind) > 7.0909) && (double(t_ind) <= 8.2727) )
          {
             t_t = double(t_ind)-7.0909;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+28]+t_t*
                        (cub_coeff[ind*N_time_points*4+28+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+28+2]+t_t*
                             cub_coeff[ind*N_time_points*4+28+3])));
          }
          else if ( (double(t_ind) > 8.2727) && (double(t_ind) <= 9.4545) )
          {
             t_t = double(t_ind)-8.2727;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+32]+t_t*
                        (cub_coeff[ind*N_time_points*4+32+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+32+2]+t_t*
                             cub_coeff[ind*N_time_points*4+32+3])));
          }
          else if ( (double(t_ind) > 9.4545) && (double(t_ind) <=10.6364) )
          {
             t_t = double(t_ind)-9.4545;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+36]+t_t*
                        (cub_coeff[ind*N_time_points*4+36+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+36+2]+t_t*
                             cub_coeff[ind*N_time_points*4+36+3])));
          }
          else if ( (double(t_ind) >10.6364) && (double(t_ind) <=11.8182) )
          {
             t_t = double(t_ind)-10.6364;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+40]+t_t*
                        (cub_coeff[ind*N_time_points*4+40+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+40+2]+t_t*
                             cub_coeff[ind*N_time_points*4+40+3])));
          }
          else if ( (double(t_ind) >11.8182) && (double(t_ind) <= 13.0  ) )
          {
             t_t = double(t_ind)-11.8182;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+44]+t_t*
                        (cub_coeff[ind*N_time_points*4+44+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+44+2]+t_t*
                             cub_coeff[ind*N_time_points*4+44+3])));
          }
          else if (double(t_ind) > 13.0  )
          {
             t_t = double(t_ind)-13.0;
             x_spline.push_back(cub_coeff[ind*N_time_points*4+48]+t_t*
                        (cub_coeff[ind*N_time_points*4+48+1]+t_t*
                        (cub_coeff[ind*N_time_points*4+48+2]+t_t*
                             cub_coeff[ind*N_time_points*4+48+3])));
          }
     }
}
// Calculate mean and standard deviation
void calc_stats( const vector<double> &x_spline, vector<double> &mean, vector<double> &sd )
{
     double mean_temp = 0.0;
     mean_temp = (accumulate(x_spline.begin(),x_spline.end(),0.0)/double(x_spline.size()));
     mean.push_back(mean_temp);
     //Calculate standard deviation
     double sd_temp = 0.0;
     for ( int k = 0; k < x_spline.size(); k++)
     {
         sd_temp += pow((x_spline[k]-mean_temp),2.0);
     }
     sd_temp = sqrt(sd_temp/double(x_spline.size()));
     sd.push_back(sd_temp);
}
// Calculate correlation
void calc_corr( const int &N_gene, const vector<double> &x_spline, const vector<double> &mean_xd, const vector<double> &sd_xd, vector<double> &correlation )
{
   int size_vec = x_spline.size()/N_gene;
   for ( int ind_i=0; ind_i < N_gene; ind_i++)
   {
       for ( int ind_j=0; ind_j < N_gene; ind_j++)
       {
            if (ind_i != ind_j)
            {
               int start = ind_i*size_vec + 1;
               int end = ((ind_i + 1) * size_vec);
               vector <double> x_spline_i ( x_spline.begin() + start, x_spline.begin() + end );
               start = ind_j*size_vec;
               end = ((ind_j + 1) * size_vec) - 1;
               vector <double> x_spline_j ( x_spline.begin() + start, x_spline.begin() + end );
               // One can use x_spline_i or j to save memory
               vector <double> corr_vector(x_spline_i.size());
               transform(x_spline_i.begin(),x_spline_i.end(),x_spline_j.begin(),corr_vector.begin(),corr_functor(mean_xd[ind_i],mean_xd[ind_j]));
               double corr_ij;
               corr_ij = accumulate(corr_vector.begin(),corr_vector.end(),0.0)/(x_spline_i.size()*sd_xd[ind_i]*sd_xd[ind_j]);
               correlation.push_back(corr_ij);               
            }
       }
   }
} 
