#include "hip/hip_runtime.h"
# include "../common.h"
using namespace std;
# include "../lsoda/cuLsoda.hpp"
# include "../opt/opt.hpp"
# include "./search.hpp"
void random_conn( const int &gene_ind, unsigned int seed_int, vector <double> &mean, int &n_ka_temp, int &n_kd_temp, vector <int> &kavec_temp, vector <int> &kdvec_temp, vector <double> &kaval_temp, vector <double> &kdval_temp)
{
    thrust::default_random_engine r_gen(time(NULL)*seed_int);
    thrust::uniform_int_distribution<int> u2(0,RAND_MAX); 
    vector<bool> nodes_on(N_gene), free_nodes(N_gene);
    vector<int> free_nodes_vec;
    fill(nodes_on.begin(), nodes_on.end(), false);
    nodes_on[gene_ind] = true;	
    for ( int i = 0; i < n_ka_temp; i++ ) nodes_on[kavec_temp[i]] = true;	
    for ( int i = 0; i < n_kd_temp; i++ ) nodes_on[kdvec_temp[i]] = true;	
    free_nodes = nodes_on;
    free_nodes.flip();  
    for ( int j = 0; j < N_gene; j++) 
    {
        if (free_nodes[j]) free_nodes_vec.push_back(j); 
    }
    // switch_vec represents 45% probability of adding(1)/deleting(2) a connection
    // and 10% probability of doing nothing
    int sum_elems, switch_call;
    sum_elems = accumulate(nodes_on.begin(),nodes_on.end(),0);
    
    if (sum_elems == N_gene)//Only remove connections or do nothing
    {
       // switch_vec represents 90% probability of deleting(2) a connection
       // and 10% probability of doing nothing
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(2);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(9+1) + 1)];  
    } 
    else if (sum_elems == 1)
    {
       // switch_vec represents 90% probability of adding(1) a connection
       // and 10% probability of doing nothing
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(1);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(9+1) + 1)];  
    }
    else
    {
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(1);
       for (int i = 0; i < 9;i++) switch_vec.push_back(2);
       switch_vec.push_back(3);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(19+1) + 1)];  
    }
    switch(switch_call)
    {
       case 1:
              {
                 random_shuffle(free_nodes_vec.begin(), free_nodes_vec.end());
                 int add_conn;
                 add_conn = free_nodes_vec[rand()/(RAND_MAX/(free_nodes_vec.size()) + 1)];
                 // ka and kd values in log space so no need of fabs for guassrand()   
                 if (u2(r_gen)%2 == 0) 
                 {
                    kavec_temp.push_back(add_conn); 
                    kaval_temp.push_back((1.0/mean[add_conn]));
                    n_ka_temp = n_ka_temp + 1; 
                 }     
                 else 
                 {
                    kdvec_temp.push_back(add_conn); 
                    kdval_temp.push_back((1.0/mean[add_conn])); 
                    n_kd_temp = n_kd_temp + 1; 
                 }                
                 break;
              }
       case 2:
              {
                 if ((u2(r_gen)%2 == 0) && (n_ka_temp != 0))
                 {
                    vector<int> indices(n_ka_temp);
                    for (int ind = 0; ind < n_ka_temp; ind++) indices[ind] = ind;
                    random_shuffle(indices.begin(), indices.end());
                    int del_conn = indices[rand()/(RAND_MAX/(indices.size()) + 1)];
                    kavec_temp.erase(kavec_temp.begin()+del_conn);
                    kaval_temp.erase(kaval_temp.begin()+del_conn);
                    n_ka_temp = n_ka_temp - 1; 
                 }
                 else if (n_kd_temp !=0)
                 {
                    vector<int> indices(n_kd_temp);
                    for (int ind = 0; ind < n_kd_temp; ind++) indices[ind] = ind;
                    random_shuffle(indices.begin(), indices.end());
                    int del_conn = indices[rand()/(RAND_MAX/(indices.size()) + 1)];
                    kdvec_temp.erase(kdvec_temp.begin()+del_conn);
                    kdval_temp.erase(kdval_temp.begin()+del_conn);
                    n_kd_temp = n_kd_temp - 1; 
                 }         
                 break;
              }
       case 3:
              break;
    }
};
