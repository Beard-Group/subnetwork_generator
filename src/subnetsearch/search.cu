#include "hip/hip_runtime.h"
# include "../common.h"
using namespace std;
# include "../lsoda/cuLsoda.hpp"
# include "../opt/opt.hpp"
# include "./search.hpp"
void random_conn( const int &gene_ind, unsigned int seed_int, vector <double> &mean, int &n_ka_temp, int &n_kd_temp, vector <int> &kavec_temp, vector <int> &kdvec_temp, vector <double> &kaval_temp, vector <double> &kdval_temp)
{
    //unsigned int seed = 1;
    //cout << move_functor.hash(seed)<< endl;
    //srand(move_functor.hash(1));
    //cout << "Value of the seed" << seed_gen.generate_seed(seed_int) << endl;
    //srand(seed_int*time(NULL));
    thrust::default_random_engine r_gen(time(NULL)*seed_int);
    thrust::uniform_int_distribution<int> u2(0,RAND_MAX); 
    vector<bool> nodes_on(N_gene), free_nodes(N_gene);
    vector<int> free_nodes_vec;
    fill(nodes_on.begin(), nodes_on.end(), false);
    nodes_on[gene_ind] = true;	
    for ( int i = 0; i < n_ka_temp; i++ ) nodes_on[kavec_temp[i]] = true;	
    for ( int i = 0; i < n_kd_temp; i++ ) nodes_on[kdvec_temp[i]] = true;	
    free_nodes = nodes_on;
    free_nodes.flip();  
    /*for ( int j = 0; j < N_gene; j++) 
    {
        if (nodes_on[j]) cout << "###############  Node " << j << " is ON!  #############" << endl;
    }*/
    for ( int j = 0; j < N_gene; j++) 
    {
        if (free_nodes[j]) free_nodes_vec.push_back(j); 
    }
    // switch_vec represents 45% probability of adding(1)/deleting(2) a connection
    // and 10% probability of doing nothing
    int sum_elems, switch_call;
    sum_elems = accumulate(nodes_on.begin(),nodes_on.end(),0);
    
    if (sum_elems == N_gene)//Only remove connections or do nothing
    {
       // switch_vec represents 90% probability of deleting(2) a connection
       // and 10% probability of doing nothing
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(2);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(9+1) + 1)];  
    } 
    else if (sum_elems == 1)
    {
       // switch_vec represents 90% probability of adding(1) a connection
       // and 10% probability of doing nothing
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(1);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(9+1) + 1)];  
    }
    else
    {
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(1);
       for (int i = 0; i < 9;i++) switch_vec.push_back(2);
       switch_vec.push_back(3);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(19+1) + 1)];  
    }
    //cout << "---------- switch_call is: "<< switch_call << " -----------" << endl;
    switch(switch_call)
    {
       case 1:
              {
                 random_shuffle(free_nodes_vec.begin(), free_nodes_vec.end());
                 int add_conn;
                 add_conn = free_nodes_vec[rand()/(RAND_MAX/(free_nodes_vec.size()) + 1)];
                 // ka and kd values in log space so no need of fabs for guassrand()   
                 //if ((rand()%2 == 0) && (n_ka_temp != 0))
                 if (u2(r_gen)%2 == 0) 
                 {
                    //srand(time(NULL)+seed_int+add_conn);
                    //thrust::default_random_engine r_gen(rand());
                    //thrust::uniform_real_distribution<double> u2(-2,2); 
                    kavec_temp.push_back(add_conn); 
                    //kaval_temp.push_back(pow(10.0,u2(r_gen)));
                    kaval_temp.push_back((1.0/mean[add_conn]));
                    n_ka_temp = n_ka_temp + 1; 
                 }     
                 else 
                 {
                    //srand(time(NULL)+seed_int+add_conn);
                    //thrust::default_random_engine r_gen(rand());
                    //thrust::uniform_real_distribution<double> u2(-2,2); 
                    kdvec_temp.push_back(add_conn); 
                    //kdval_temp.push_back(pow(10.0,u2(r_gen)));
                    kdval_temp.push_back((1.0/mean[add_conn])); 
                    n_kd_temp = n_kd_temp + 1; 
                 }                
                 break;
              }
       case 2:
              {
                 if ((u2(r_gen)%2 == 0) && (n_ka_temp != 0))
                 {
                    vector<int> indices(n_ka_temp);
                    for (int ind = 0; ind < n_ka_temp; ind++) indices[ind] = ind;
                    random_shuffle(indices.begin(), indices.end());
                    int del_conn = indices[rand()/(RAND_MAX/(indices.size()) + 1)];
                    kavec_temp.erase(kavec_temp.begin()+del_conn);
                    kaval_temp.erase(kaval_temp.begin()+del_conn);
                    n_ka_temp = n_ka_temp - 1; 
                 }
                 else if (n_kd_temp !=0)
                 {
                    vector<int> indices(n_kd_temp);
                    for (int ind = 0; ind < n_kd_temp; ind++) indices[ind] = ind;
                    random_shuffle(indices.begin(), indices.end());
                    int del_conn = indices[rand()/(RAND_MAX/(indices.size()) + 1)];
                    kdvec_temp.erase(kdvec_temp.begin()+del_conn);
                    kdval_temp.erase(kdval_temp.begin()+del_conn);
                    n_kd_temp = n_kd_temp - 1; 
                 }         
                 break;
              }
       case 3:
              break;
    }
};
/*void biased_select( const int &gene_ind, unsigned int seed_int, vector <double> &mean, const state_type &corr, int &n_ka_temp, int &n_kd_temp, vector <int> &kavec_temp, vector <int> &kdvec_temp, state_type &kaval_temp, state_type &kdval_temp)
{
    thrust::default_random_engine r_gen(time(NULL)*seed_int);
    thrust::uniform_int_distribution<int> u2(0,RAND_MAX); 
    vector<bool> nodes_on(N_gene), free_nodes(N_gene);
    vector<int> free_nodes_vec;
    fill(nodes_on.begin(), nodes_on.end(), false);
    nodes_on[gene_ind] = true;	
    for ( int i = 0; i < n_ka_temp; i++ ) nodes_on[kavec_temp[i]] = true;	
    for ( int i = 0; i < n_kd_temp; i++ ) nodes_on[kdvec_temp[i]] = true;	
    free_nodes = nodes_on;
    free_nodes.flip();  
    for ( int j = 0; j < N_gene; j++) 
    {
        if (free_nodes[j]) free_nodes_vec.push_back(j); 
    }
    // switch_vec represents 45% probability of adding(1)/deleting(2) a connection
    // and 10% probability of doing nothing
    int sum_elems, switch_call;
    sum_elems = accumulate(nodes_on.begin(),nodes_on.end(),0);
    
    if (sum_elems == N_gene)//Only remove connections or do nothing
    {
       // switch_vec represents 90% probability of deleting(2) a connection
       // and 10% probability of doing nothing
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(2);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(9+1) + 1)];  
    } 
    else if (sum_elems == 1)
    {
       // switch_vec represents 90% probability of adding(1) a connection
       // and 10% probability of doing nothing
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(1);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(9+1) + 1)];  
    }
    else
    {
       vector<int> switch_vec;
       for (int i = 0; i < 9;i++) switch_vec.push_back(1);
       for (int i = 0; i < 9;i++) switch_vec.push_back(2);
       switch_vec.push_back(3);
       switch_vec.push_back(3);
       random_shuffle(switch_vec.begin(), switch_vec.end());
       switch_call = switch_vec[rand()/(RAND_MAX/(19+1) + 1)];  
    }
    //cout << "---------- switch_call is: "<< switch_call << " -----------" << endl;
    switch(switch_call)
    {
       case 1:
              {
                 //vector <double> free_nodes_weight(free_nodes_vec.size());
                 vector <double> free_nodes_weight(free_nodes_vec.size());
                 // assign interval as indices to the free_node vector
                 //vector <double> indices; 
                 vector <int> indices(free_nodes_vec.size()); 
                 // Determine the weights of each interval
                 //indices.push_back(0.0);
                 for ( int ind = 0; ind < free_nodes_vec.size(); ind++)
                 {
                     //free_nodes_weight[ind] = fabs(corr[gene_ind*(N_gene-1)+free_nodes_vec[ind]]);  
                     free_nodes_weight[ind]= fabs(corr[gene_ind*(N_gene-1)+free_nodes_vec[ind]]);  
                     //indices.push_back(double(ind+1));
                     indices[ind]=ind;
                 }
                 //piecewise_constant_distribution<>dist(indices.begin(), indices.end(), free_nodes_weight.begin());
                 //mt19937 gen(time(0));
                 //clock_t time_seed = clock();
                 //unsigned int seed = rand()*time_seed;
                 //thrust::default_random_engine rng(seed);
                 //vector <double> inv_free_nodes_weight(free_nodes_vec.size()); 
                 //thrust::transform(free_nodes_weight.begin(),free_nodes_weight.end(),inv_free_nodes_weight.begin(),invert_functor());
                 thrust::sort_by_key(free_nodes_weight.begin(),free_nodes_weight.end(),indices.begin(),thrust::greater<double>());
                 thrust::random::experimental::normal_distribution<double> u01(0.0,3.0);
                 // Then determine the free node index using correlation
                 //int add_conn_ind = int(dist(gen));
                 int add_conn_ind = (int)floor(fabs(u01(r_gen)));
                 int add_conn = free_nodes_vec[indices[add_conn_ind]];
                 if ( corr[gene_ind*(N_gene-1)+free_nodes_vec[add_conn_ind]] > 0.0 ) 
                 {
                    kavec_temp.push_back(add_conn); 
                    //kaval_temp.push_back(guassrand());
                    kaval_temp.push_back(mean[add_conn]);
                    n_ka_temp = n_ka_temp + 1; 
                 }     
                 else
                 {
                    kdvec_temp.push_back(add_conn); 
                    //kdval_temp.push_back(guassrand()); 
                    kdval_temp.push_back(mean[add_conn]); 
                    n_kd_temp = n_kd_temp + 1; 
                 }                
                 //add_conn = free_nodes_vec[int(dist(gen))];
                 // Use that to determine the free node vector
                 // Add a connection from the free nodes vector
                 break;
              }
       case 2:
              {
                 //if ((rand()%2 == 0) && (n_ka_temp != 0))
                 if ((u2(r_gen)%2 == 0) && (n_ka_temp != 0)) 
                 {
                    vector<int> indices(n_ka_temp);
                    for (int ind = 0; ind < n_ka_temp; ind++) indices[ind] = ind;
                    random_shuffle(indices.begin(), indices.end());
                    int del_conn = indices[rand()/(RAND_MAX/(indices.size()) + 1)];
                    kavec_temp.erase(kavec_temp.begin()+del_conn);
                    kaval_temp.erase(kaval_temp.begin()+del_conn);
                    n_ka_temp = n_ka_temp - 1; 
                 }
                 else if (n_kd_temp !=0)
                 {
                    vector<int> indices(n_kd_temp);
                    for (int ind = 0; ind < n_kd_temp; ind++) indices[ind] = ind;
                    random_shuffle(indices.begin(), indices.end());
                    int del_conn = indices[rand()/(RAND_MAX/(indices.size()) + 1)];
                    kdvec_temp.erase(kdvec_temp.begin()+del_conn);
                    kdval_temp.erase(kdval_temp.begin()+del_conn);
                    n_kd_temp = n_kd_temp - 1; 
                 }         
                 break;
              }
       case 3:
              break;
    }
};*/
