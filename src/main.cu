#include "hip/hip_runtime.h"
#include "./common.h"
#include "./io/io.hpp"
#include "./spline/spline.hpp"
#include "./subnetsearch/search.hpp"
#include "./lsoda/cuLsoda.hpp"
#include "./opt/opt.hpp"

using namespace std;
//
int main(void)    
{  
   hipSetDevice(2);
   cout << "----Starting simulation----" << endl; 	
   cout << "----Reading in data----" << endl; 	
   vector <double> t_d(N_time_points);
   vector <double> x_d(N_gene*N_time_points);
   //vector <double> sd_d(N_gene*N_time_points);
   read_data (N_time_points, N_gene, t_d, x_d);
   cout << "----Setting up spline co-efficients----" << endl; 	
   vector <double> deriv_time(N_time_points*N_gene);
   //cout << "  Setting up co-fficients of cubic spline" << endl;
   for ( int i = 0 ; i < N_gene ; i++ )
   {
       spline_deriv ( i, N_time_points, t_d, x_d, deriv_time);
   }
   // Set up the co-effients matrix 
   vector <double> cub_coeff_spline (N_gene*N_time_points*4);
   spline_coeff ( N_gene, N_time_points, t_d, x_d, deriv_time, cub_coeff_spline);
   // Determine mean, standard deviation, and correlation statistics
   //vector <double> mean_xd, sd_xd, x_spline ;
   vector <double> mean_xd, sd_xd;
   for ( int ind = 0; ind < N_gene; ind++)
   { 
       //x_spline represented in terms of dt = 1. Used to determine mean, sd, and correlation
       // N=10 case dt=1 for the data anyway so not necessary
       vector <double> x_spline_temp(N_time_points);
       copy( x_d.begin()+ind*N_time_points, x_d.begin()+(ind+1)*N_time_points, x_spline_temp.begin());
       //calc_spline( x_d, t_d, cub_coeff_spline, ind, x_spline_temp);
       calc_stats( x_spline_temp, mean_xd, sd_xd);
       vector<double>().swap(x_spline_temp);
       //cout << "Value of mean is: " << mean_xd[ind] << endl;
       //cout << "Value of sd is: " << sd_xd[ind] << endl;
       //x_spline.insert(x_spline.end(),x_spline_temp.begin(),x_spline_temp.end());
   }
   cout << "----END spline setup----" << endl; 	
   //vector <double> correlation;
   //calc_corr(N_gene, x_d, mean_xd, sd_xd, correlation);
   //Error
   double E0 = 3.0E-03;
   //Total simulation time
   double sim_time = 0.0;
   //const int N_tot = 20;
   for (int id_gene = 644; id_gene < 645; id_gene++)
   {
      const int gene_ind = id_gene;
      vector <double> act_vec, inh_vec;
      vector <int> subnet_size;
      bool r0_out = false;
      bool ea_out = false;
      bool d_out = false;
      bool nka_out = false;
      bool nkd_out = false;
      bool kavec_out = false;
      bool kdvec_out = false;
      bool kaval_out = false;
      bool kdval_out = false;
      char r0_out_file[32] = ".r0";
      char ea_out_file[32] = ".ea";
      char d_out_file[32] = ".d";
      char nka_out_file[32] = ".nka";
      char nkd_out_file[32] = ".nkd";
      char kavec_out_file[32] = ".kavec";
      char kdvec_out_file[32] = ".kdvec";
      char kaval_out_file[32] = ".kaval";
      char kdval_out_file[32] = ".kdval";
      char r0_index_string[64];
      char r0_out_path[96] = "../out/N1000/";
      char ea_index_string[64];
      char ea_out_path[96] = "../out/N1000/";
      char d_index_string[64];
      char d_out_path[96] = "../out/N1000/";
      char nka_index_string[64];
      char nka_out_path[96] = "../out/N1000/";
      char nkd_index_string[64];
      char nkd_out_path[96] = "../out/N1000/";
      char kavec_index_string[64];
      char kavec_out_path[96] = "../out/N1000/";
      char kdvec_index_string[64];
      char kdvec_out_path[96] = "../out/N1000/";
      char kaval_index_string[64];
      char kaval_out_path[96] = "../out/N1000/";
      char kdval_index_string[64];
      char kdval_out_path[96] = "../out/N1000/";
      if (!r0_out)
      {
         r0_out = true;
         sprintf(r0_index_string, "%d", gene_ind+1);
         strcat(r0_index_string,r0_out_file);
         strcat(r0_out_path,r0_index_string);
      }
      string out_r0(r0_out_path);
      if (!ea_out)
      {
         ea_out = true;
         sprintf(ea_index_string, "%d", gene_ind+1);
         strcat(ea_index_string,ea_out_file);
         strcat(ea_out_path,ea_index_string);
      }
      string out_ea(ea_out_path);
      if (!d_out)
      {
         d_out = true;
         sprintf(d_index_string, "%d", gene_ind+1);
         strcat(d_index_string,d_out_file);
         strcat(d_out_path,d_index_string);
      }
      string out_d(d_out_path);
      if (!nka_out)
      {
         nka_out = true;
         sprintf(nka_index_string, "%d", gene_ind+1);
         strcat(nka_index_string,nka_out_file);
         strcat(nka_out_path,nka_index_string);
      }
      string out_nka(nka_out_path);
      if (!nkd_out)
      {
         nkd_out = true;
         sprintf(nkd_index_string, "%d", gene_ind+1);
         strcat(nkd_index_string,nkd_out_file);
         strcat(nkd_out_path,nkd_index_string);
      }
      string out_nkd(nkd_out_path);
      if (!kavec_out)
      {
         kavec_out = true;
         sprintf(kavec_index_string, "%d", gene_ind+1);
         strcat(kavec_index_string,kavec_out_file);
         strcat(kavec_out_path,kavec_index_string);
      }
      string out_kavec(kavec_out_path);
      if (!kdvec_out)
      {
         kdvec_out = true;
         sprintf(kdvec_index_string, "%d", gene_ind+1);
         strcat(kdvec_index_string,kdvec_out_file);
         strcat(kdvec_out_path,kdvec_index_string);
      }
      string out_kdvec(kdvec_out_path);
      if (!kaval_out)
      {
         kaval_out = true;
         sprintf(kaval_index_string, "%d", gene_ind+1);
         strcat(kaval_index_string,kaval_out_file);
         strcat(kaval_out_path,kaval_index_string);
      }
      string out_kaval(kaval_out_path);
      if (!kdval_out)
      {
         kdval_out = true;
         sprintf(kdval_index_string, "%d", gene_ind+1);
         strcat(kdval_index_string,kdval_out_file);
         strcat(kdval_out_path,kdval_index_string);
      }
      string out_kdval(kdval_out_path);
      int acc_subnet = 0;
      int subnet_att = 0;
      //int net_count = 0;
      //vector <int> nka_vec_acc, nkd_vec_acc, kavec_acc, kdvec_acc;
      //read_in_subnet ( gene_ind, nka_vec_acc, nkd_vec_acc, kavec_acc, kdvec_acc);
      //const int N_trial_net = nka_vec_acc.size();
      //boost::timer timer;
      double proc_time = 0.0;
      cout << "------------ Start for gene " << gene_ind+1 << "  ------" << endl;
      while ( (acc_subnet < N_att) && (subnet_att<50) )
      //for (int k = 0; k < N_tot; k++)
      {
         if (subnet_att <= 500) E0=5.0E-03;
         else E0=7.0E-03;
         cout << "------------ Attempt number " << subnet_att << "  ------" << endl;
         //else if ((subnet_att>250) && (subnet_att<=750)) E0=5.0E-03;
         //else if ((subnet_att>750) && (subnet_att<=1000)) E0=7.5E-03;
         //else if (subnet_att>1000) E0=1.0E-02;
         //else E0 = 1.0E-02;
         subnet_att++;
         int *n_ka = (int*)malloc(sizeof(int)*probSize);
         int *n_kd = (int*)malloc(sizeof(int)*probSize);
         int *ka_start = (int*)malloc(sizeof(int)*probSize);
         int *kd_start = (int*)malloc(sizeof(int)*probSize);
         int ka_ind = 0;
         int kd_ind = 0;
         vector <int> temp_ka_vec, temp_kd_vec;
         vector <double> temp_ka_val, temp_kd_val;
         srand(time(NULL));
         // generate integers between 50 to 200
         thrust::default_random_engine rng_gen(time(NULL)*rand());
         thrust::uniform_int_distribution<int> u_l(50,200); 
         for ( unsigned int ind = 0; ind < probSize; ind++ )
         {
            int n_ka_temp = 0;
            int n_kd_temp = 0;
            // How to declare kavec_temp and kd_vec_temp
            vector <int> kavec_temp, kdvec_temp;
            vector <double> kaval_temp, kdval_temp;
            //int conn_limit = 50 + int ((150*rand())/(RAND_MAX+1.0));
            /*if (net_count < N_trial_net)
            {
                *(n_ka+ind) = nka_vec_acc[net_count];
                *(n_kd+ind) = nkd_vec_acc[net_count];
                temp_ka_vec.insert(temp_ka_vec.end(),kavec_acc.begin()+ka_ind,kavec_acc.begin()+ka_ind+nka_vec_acc[net_count]); 
                temp_kd_vec.insert(temp_kd_vec.end(),kdvec_acc.begin()+kd_ind,kdvec_acc.begin()+kd_ind+nkd_vec_acc[net_count]);
                for (int i = 0; i < nka_vec_acc[net_count]; i++)
                    temp_ka_val.push_back(mean_xd[kavec_acc[ka_ind+i]]); 
                for (int i = 0; i < nkd_vec_acc[net_count]; i++)
                    temp_kd_val.push_back(mean_xd[kdvec_acc[kd_ind+i]]); 
                *(ka_start+ind) = ka_ind;
                *(kd_start+ind) = kd_ind;
                ka_ind += nka_vec_acc[net_count];
                kd_ind += nkd_vec_acc[net_count];
                net_count++;
            }
            else 
            {*/
            for (unsigned int k = 0; k < u_l(rng_gen); k++) 
                random_conn(gene_ind, k+ind, mean_xd, n_ka_temp, n_kd_temp, kavec_temp, kdvec_temp, kaval_temp, kdval_temp);
            *(n_ka+ind) = n_ka_temp;
            *(n_kd+ind) = n_kd_temp;
            *(ka_start+ind) = ka_ind;
            *(kd_start+ind) = kd_ind;
            ka_ind += n_ka_temp;
            kd_ind += n_kd_temp;
            temp_ka_vec.insert(temp_ka_vec.end(),kavec_temp.begin(),kavec_temp.end()); 
            temp_kd_vec.insert(temp_kd_vec.end(),kdvec_temp.begin(),kdvec_temp.end()); 
            temp_ka_val.insert(temp_ka_val.end(),kaval_temp.begin(),kaval_temp.end()); 
            temp_kd_val.insert(temp_kd_val.end(),kdval_temp.begin(),kdval_temp.end());
            //}
            vector <int>().swap(kavec_temp);
            vector <int>().swap(kdvec_temp);
            vector <double>().swap(kaval_temp);
            vector <double>().swap(kdval_temp);
         }
         //
         const int size_ka = temp_ka_vec.size();
         const int size_kd = temp_kd_vec.size();
         //const int N_tot = 1;
         host_type error_sim_h;
         //for (int N_itr = 0; N_itr < N_tot; N_itr++)
         //{
         int *ka_vec = (int*)malloc(sizeof(int)*size_ka);
         double *ka_val = (double*)malloc(sizeof(double)*size_ka);
         int l = 0;
         while ( l < size_ka )
         {
             *(ka_vec+l) = temp_ka_vec[l];
             *(ka_val+l) = temp_ka_val[l];
             l++;
         } 
         l = 0;
         int *kd_vec = (int*)malloc(sizeof(int)*size_kd);
         double *kd_val = (double*)malloc(sizeof(double)*size_kd);
         while ( l < size_kd )
         {
             *(kd_vec+l) = temp_kd_vec[l];
             *(kd_val+l) = temp_kd_val[l];
             l++;
         } 
         double *r0 = (double*)malloc(sizeof(double)*probSize);
         double *d = (double*)malloc(sizeof(double)*probSize);
         double *ea = (double*)malloc(sizeof(double)*probSize);
         srand(time(NULL));
         for ( int i = 0; i < probSize; i++)
         {
               // *(r0+i) = pow(10.0,guassrand());
               // *(d+i) = pow(10.0,guassrand());
               // *(ea+i) = pow(10.0,guassrand());
               *(r0+i) = mean_xd[gene_ind];
               *(d+i) = 1.0;
               *(ea+i) = 1.0;
         }
         double cub_coeff[N_gene*N_time_points*4];
         for ( int i = 0; i < cub_coeff_spline.size(); i++) cub_coeff[i] = cub_coeff_spline[i];
         myFex fex;
         fex.set_r0(r0);
         fex.set_d(d);
         fex.set_ea(ea);
         fex.set_n_ka(n_ka);
         fex.set_ka_vec(ka_vec,size_ka);
         fex.set_ka_start(ka_start);
         fex.set_ka_val(ka_val,size_ka);
         fex.set_n_kd(n_kd);
         fex.set_kd_start(kd_start);
         fex.set_kd_vec(kd_vec,size_kd);
         fex.set_kd_val(kd_val,size_kd);
         int size_coeff = N_gene*N_time_points*4;
         fex.set_coeff(cub_coeff, size_coeff); 
         myJex jex;
         //timer.restart();
         state_type error_sim_d(probSize);
         // Integrating using LSODE
         // Integrate ODEs
         //timer.restart();
         integrate_lsoda_ode (gene_ind, x_d, t_d, mean_xd[gene_ind], fex, jex, error_sim_d);
         // MC Simulation
         //cout << "ODE END" << endl;
         MC_sim(gene_ind, x_d, t_d, mean_xd[gene_ind], n_ka, n_kd, size_ka, size_kd, fex, jex, error_sim_d);
         //cout << "MC END" << endl;
         host_type ka_val_h(size_ka), kd_val_h(size_kd);
         //thrust::host_vector<int> ka_vec_h, kd_vec_h;
         //fex.get_kavec_vec(ka_vec_h,size_ka);
         //cout << "Count Begin" << endl;
         host_type r0_mc(probSize), d_mc(probSize), ea_mc(probSize);
         thrust::device_ptr<double> r0_mc_d, d_mc_d, ea_mc_d;
         fex.get_r0_ptr(r0_mc_d);
         fex.get_ea_ptr(ea_mc_d);
         fex.get_d_ptr(d_mc_d);
         thrust::copy(r0_mc_d, r0_mc_d+probSize, r0_mc.begin()); 
         thrust::copy(ea_mc_d, ea_mc_d+probSize, ea_mc.begin()); 
         thrust::copy(d_mc_d, d_mc_d+probSize, d_mc.begin()); 
         fex.get_kaval_vec(ka_val_h,size_ka);
         //fex.get_kdvec_vec(kd_vec_h,size_kd);
         //cout << "Count Begin" << endl;
         fex.get_kdval_vec(kd_val_h,size_kd);
         //proc_time = timer.elapsed()/(double(N_tot));
         //proc_time += timer.elapsed();
         //cout << "MC END" << endl;
         error_sim_h = error_sim_d;
         //cout << "Count Begin" << endl;
         //thrust::sort(error_sim_h.begin(),error_sim_h.end()); 
         //cout << "Value of Simulation error is: " << error_sim_h[0] << endl;
         //cout << "Value of Simulation error is: " << error_sim_h[1] << endl;
         //cout << "Value of Simulation error is: " << error_sim_h[2] << endl;
         fex.set_r0_free();
         fex.set_d_free();
         fex.set_ea_free();
         fex.set_n_ka_free();
         fex.set_ka_vec_free();
         fex.set_ka_start_free();
         fex.set_ka_val_free();
         fex.set_n_kd_free();
         fex.set_kd_start_free();
         fex.set_kd_vec_free();
         fex.set_kd_val_free();
         fex.set_coeff_free(); 
         //proc_time = timer.elapsed();
         //cout << "Count Begin" << endl;
         for ( int ind = 0; ind < probSize; ind++)
         {
             if ( error_sim_h[ind] < E0 ) 
             //if ( (error_ode < E_0) && ((*(n_ka+ind)+*(n_kd_ind)>1) && (*(n_ka+ind)+*(n_kd+ind)<6)) )
             {
                    bool acc_sub = false; 
                    int act_count = 0;
                    int inh_count = 0;
                    int start_ka = *(ka_start+ind);
                    int end_ka = *(ka_start+ind)+*(n_ka+ind);
                    int start_kd = *(kd_start+ind);
                    int end_kd = *(kd_start+ind)+*(n_kd+ind);
                    double r0_ = r0_mc[ind];
                    double ea_ = ea_mc[ind];
                    double d_ = d_mc[ind];
                    output_data ( gene_ind, r0_, ea_, d_, out_r0, out_ea, out_d, start_ka, end_ka, start_kd, end_kd, temp_ka_vec, temp_kd_vec, ka_val_h, kd_val_h, out_nka, out_nkd, out_kavec, out_kdvec, out_kaval, out_kdval);
                    //cout << "######" << " Value of error is: " << error_sim_h[ind] << "#######" << endl;
                    for ( int k = *(ka_start+ind); k < (*(ka_start+ind)+*(n_ka+ind)); k++) 
                    {
                       if ( ka_val_h[k] > 0.05 ) 
                       {  
                          act_vec.push_back(temp_ka_vec[k]);   
                          acc_sub = true;
                          act_count++;
                       } 
                    }
                    for ( int k = *(kd_start+ind); k < (*(kd_start+ind)+*(n_kd+ind)); k++) 
                    {
                       if ( kd_val_h[k] > 0.05 ) 
                       {   
                          inh_vec.push_back(temp_kd_vec[k]);   
                          acc_sub = true;
                          inh_count++;
                       }
                    }
                    //act_vec.insert(act_vec.end(),temp_ka_vec.begin()+*(ka_start+ind),temp_ka_vec.begin()+*(ka_start+ind)+*(n_ka+ind)); 
                    //inh_vec.insert(inh_vec.end(),temp_kd_vec.begin()+*(kd_start+ind),temp_kd_vec.begin()+*(kd_start+ind)+*(n_kd+ind)); 
                    if (acc_sub) 
                    {
                       acc_subnet++;
                       subnet_size.push_back(act_count+inh_count);
                       cout << "-------- Acceptable subnet number: " << acc_subnet << "  ------------" << endl; 
                       cout << "-------- Size of subnetwork: " << act_count+inh_count << "  ------------" << endl; 
                       cout << "-------- Value of error is : " << error_sim_h[ind] << "  ------------" << endl; 
                       cout << "-------- Number of attempts made till now: " << subnet_att << "  ------------" << endl; 
                    }
             } 
         }
         error_sim_h.clear();
         error_sim_h.shrink_to_fit();
         vector <int>().swap(temp_ka_vec);
         vector <int>().swap(temp_kd_vec);
         vector <double>().swap(temp_ka_val);
         vector <double>().swap(temp_kd_val);
         free(r0);
         free(d);
         free(ea);
         free(n_ka);
         free(n_kd);
         free(ka_start);
         free(kd_start);
         free(ka_vec);
         free(kd_vec);
         free(ka_val);
         free(kd_val);
      }
      char out_file[32] = ".out";
      char index_string[64];
      sprintf(index_string, "%d", gene_ind+1);
      strcat(index_string,out_file);
      char out_path[96] = "../out/N1000/";
      //char out_path[96] = "/scratch/raghut/GPU_g1g2/";
      strcat(out_path,index_string);
      ofstream fileout;
      fileout.open( out_path, ios_base::binary|ios_base::app|ios_base::out );
      if (fileout.is_open())
      {
         cout << "File open successful!" << endl;
         fileout << "------------ START FOR ACTIVATORS --------------" << endl;
         for ( int ind_con = 0; ind_con < N_gene; ind_con++ )
         {
            int act_count = count(act_vec.begin(),act_vec.end(),ind_con);
            //if (double(act_count) > double(0.20)*(double(acc_subnet))) 
            fileout << setw(3) << ind_con+1 << "   " << setw(3) << act_count << endl; 
         }
         fileout << "-------------- END FOR ACTIVATORS --------------" << endl;
         for ( int ind_con = 0; ind_con < N_gene; ind_con++ )
         {
            int inh_count = count(inh_vec.begin(),inh_vec.end(),ind_con); 
            //if ( double(inh_count) > double(0.20)*(double(acc_subnet))) 
            fileout << setw(3) << ind_con+1 << "   " << setw(3) << inh_count << endl; 
            //cout << "Number of times gene " << ind_con+1 << " appears as an inhibitor is: " << inh_count << endl; 
         }
         fileout << "------------ Size of subnetworks START----------- " << endl;
         for ( int ind_con = 0; ind_con < 20; ind_con++ )
         {
            int size_count = count(subnet_size.begin(),subnet_size.end(),ind_con); 
            //if ( double(inh_count) > double(0.20)*(double(acc_subnet))) 
            fileout << setw(3) << ind_con << "   " << setw(3) << size_count << endl; 
            //cout << "Number of times gene " << ind_con+1 << " appears as an inhibitor is: " << inh_count << endl; 
         }
         fileout << "------------ Size of subnetworks END----------- " << endl;
         fileout << "------------ Time taken for simulation for gene " << gene_ind+1 <<
          "  is: " << setw(18) << setprecision(8) << proc_time << " --------------" << endl;
         fileout << "------------ Number of acceptable subnetworks for gene " << gene_ind+1 <<
          " is: " << acc_subnet << " --------------" << endl;
         fileout << "------------ Number of attempts for gene " << gene_ind+1 <<
          " is: " << subnet_att*probSize << " --------------" << endl;
         fileout.close();
      }
      else cout << " open() failed" << endl;
      sim_time = sim_time + proc_time;
      vector <double>().swap(act_vec);
      vector <double>().swap(inh_vec);
      vector <int>().swap(subnet_size);
      cout << "------------ Time taken for simulation for gene " << gene_ind+1 <<
          "  is: " << setw(18) << setprecision(8) << proc_time << " --------------" << endl;
      cout << "------------ End for gene " << gene_ind+1 << "  ------" << endl;
   }
   //for ( int i = 0 ; i < error_sim_h.size(); i++)cout << "Value of Simulation error is: " << error_sim_h[i] << endl;
   /*cout << "Value of Simulation error is: " << error_sim_h[1] << endl;
   cout << "Value of Simulation error is: " << error_sim_h[2] << endl;
   cout << "Value of Simulation error is: " << error_sim_h[3] << endl;
   cout << "Value of Simulation error is: " << error_sim_h[1599] << endl;
   cout << "Value of Simulation error is: " << error_sim_h[1598] << endl;
   cout << "Value of Simulation error is: " << error_sim_h[1597] << endl;
   cout << "Value of Simulation error is: " << error_sim_h[1596] << endl;*/
   //
   //cout << "------------ Total Simulation time " << sim_time << "  ------" << endl;
   return 0;
} /* MAIN */
