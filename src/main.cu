#include "hip/hip_runtime.h"
#include "./common.h"
#include "./io/io.hpp"
#include "./spline/spline.hpp"
#include "./subnetsearch/search.hpp"
#include "./lsoda/cuLsoda.hpp"
#include "./opt/opt.hpp"

#include <sstream>
using namespace std;

// Global constants, single point of maintenance
const char * const output_path_c = "../out/N1000_test/";

void setup_spline(vector<double>& t_d_in, vector<double> x_d_in,
                  vector<double>& cub_coeff_spline_out, vector<double>& mean_xd_out);

void setup_output(int gene_ind_n, vector<string>& out_files_out);

int main(void)
{
    hipSetDevice(2);
    cout << "----Starting simulation----" << endl;
    cout << "----Reading in data----" << endl;
    vector <double> t_d(N_time_points);
    vector <double> x_d(N_gene*N_time_points);
    read_data (N_time_points, N_gene, t_d, x_d);
    vector <double> cub_coeff_spline (N_gene*N_time_points*4);
    vector<double> mean_xd;
    setup_spline(t_d, x_d, cub_coeff_spline, mean_xd);
    //Error
    double E0 = 3.0E-03;
    for (int id_gene = 644; id_gene < 645; id_gene++) {
        const int gene_ind = id_gene;
        vector <double> act_vec, inh_vec;
        vector <int> subnet_size;
        // Set up output file paths for this gene
        bool created_paths = false; // Why do we need this check? Paths only created once per gene index anyway.
        vector<string> out_files;
        if (!created_paths) {
            created_paths = true;
            setup_output(gene_ind, out_files);
        }
        // END DECLARATIONS
        int acc_subnet = 0; // count of accepted subnetworks
        int subnet_att = 0; // attempts
        cout << "------------ Start for gene " << gene_ind+1 << "  ------" << endl;
        while ( (acc_subnet < N_att) && (subnet_att<50) ) {
            if (subnet_att <= 500) E0=5.0E-03;
            else E0=7.0E-03;
            cout << "------------ Attempt number " << subnet_att << "  ------" << endl;
            subnet_att++;
            int *n_ka = (int*)malloc(sizeof(int)*probSize);
            int *n_kd = (int*)malloc(sizeof(int)*probSize);
            int *ka_start = (int*)malloc(sizeof(int)*probSize);
            int *kd_start = (int*)malloc(sizeof(int)*probSize);
            int ka_ind = 0;
            int kd_ind = 0;
            vector <int> temp_ka_vec, temp_kd_vec;
            vector <double> temp_ka_val, temp_kd_val;
            srand(time(NULL));
            // generate integers between 50 to 200
            thrust::default_random_engine rng_gen(time(NULL)*rand());
            thrust::uniform_int_distribution<int> u_l(50,200);
            // Problem size loop, 1600 subnetworks
            for ( unsigned int ind = 0; ind < probSize; ind++ ) {
                int n_ka_temp = 0;
                int n_kd_temp = 0;
                // How to declare kavec_temp and kd_vec_temp
                vector <int> kavec_temp, kdvec_temp;
                vector <double> kaval_temp, kdval_temp;
                for (unsigned int k = 0; k < u_l(rng_gen); k++)
                    random_conn(gene_ind, k+ind, mean_xd, n_ka_temp, n_kd_temp, kavec_temp,
                                kdvec_temp, kaval_temp, kdval_temp); // subnetsearch subfolder
                // Parameters
                *(n_ka+ind) = n_ka_temp; // n_ka number of activating genes
                *(n_kd+ind) = n_kd_temp; // n_kd number of deactivating genes
                *(ka_start+ind) = ka_ind;
                *(kd_start+ind) = kd_ind;
                ka_ind += n_ka_temp;
                kd_ind += n_kd_temp;
                // Here we select which are the activators and deactivators
                temp_ka_vec.insert(temp_ka_vec.end(),kavec_temp.begin(),kavec_temp.end());
                temp_kd_vec.insert(temp_kd_vec.end(),kdvec_temp.begin(),kdvec_temp.end());
                // The corresponding values go in these two
                temp_ka_val.insert(temp_ka_val.end(),kaval_temp.begin(),kaval_temp.end());
                temp_kd_val.insert(temp_kd_val.end(),kdval_temp.begin(),kdval_temp.end());
                vector <int>().swap(kavec_temp);
                vector <int>().swap(kdvec_temp);
                vector <double>().swap(kaval_temp);
                vector <double>().swap(kdval_temp);
            }
            //
            const int size_ka = temp_ka_vec.size();
            const int size_kd = temp_kd_vec.size();
            host_type error_sim_h;
            int *ka_vec = (int*)malloc(sizeof(int)*size_ka);
            double *ka_val = (double*)malloc(sizeof(double)*size_ka);
            int l = 0;
            while ( l < size_ka ) {
                *(ka_vec+l) = temp_ka_vec[l];
                *(ka_val+l) = temp_ka_val[l];
                l++;
            }
            l = 0;
            int *kd_vec = (int*)malloc(sizeof(int)*size_kd);
            double *kd_val = (double*)malloc(sizeof(double)*size_kd);
            while ( l < size_kd ) {
                *(kd_vec+l) = temp_kd_vec[l];
                *(kd_val+l) = temp_kd_val[l];
                l++;
            }
            double *r0 = (double*)malloc(sizeof(double)*probSize);
            double *d = (double*)malloc(sizeof(double)*probSize);
            double *ea = (double*)malloc(sizeof(double)*probSize);
            srand(time(NULL));
            for ( int i = 0; i < probSize; i++) {
                *(r0+i) = mean_xd[gene_ind];
                *(d+i) = 1.0;
                *(ea+i) = 1.0;
            }
            double cub_coeff[N_gene*N_time_points*4];
            for ( int i = 0; i < cub_coeff_spline.size();
                    i++) cub_coeff[i] = cub_coeff_spline[i];
            myFex fex; // ./lsoda/cuLSODA.hpp
            fex.set_r0(r0);
            fex.set_d(d);
            fex.set_ea(ea);
            fex.set_n_ka(n_ka);
            fex.set_ka_vec(ka_vec,size_ka);
            fex.set_ka_start(ka_start);
            fex.set_ka_val(ka_val,size_ka);
            fex.set_n_kd(n_kd);
            fex.set_kd_start(kd_start);
            fex.set_kd_vec(kd_vec,size_kd);
            fex.set_kd_val(kd_val,size_kd);
            int size_coeff = N_gene*N_time_points*4;
            fex.set_coeff(cub_coeff, size_coeff);
            myJex jex;
            state_type error_sim_d(probSize);
            // Integrate ODEs, gives us an error value
            integrate_lsoda_ode (gene_ind, x_d, t_d, mean_xd[gene_ind], fex, jex,
                                 error_sim_d);
            // MC Simulation, changes parameters, generates new error value
            MC_sim(gene_ind, x_d, t_d, mean_xd[gene_ind], n_ka, n_kd, size_ka, size_kd, fex,
                   jex, error_sim_d);
            host_type ka_val_h(size_ka), kd_val_h(size_kd);
            host_type r0_mc(probSize), d_mc(probSize), ea_mc(probSize);
            thrust::device_ptr<double> r0_mc_d, d_mc_d, ea_mc_d;
            fex.get_r0_ptr(r0_mc_d);
            fex.get_ea_ptr(ea_mc_d);
            fex.get_d_ptr(d_mc_d);
            // Saving the error values and param values from device to host
            thrust::copy(r0_mc_d, r0_mc_d+probSize, r0_mc.begin());
            thrust::copy(ea_mc_d, ea_mc_d+probSize, ea_mc.begin());
            thrust::copy(d_mc_d, d_mc_d+probSize, d_mc.begin());
            fex.get_kaval_vec(ka_val_h,size_ka);
            fex.get_kdval_vec(kd_val_h,size_kd);
            error_sim_h = error_sim_d;
            fex.set_r0_free();
            fex.set_d_free();
            fex.set_ea_free();
            fex.set_n_ka_free();
            fex.set_ka_vec_free();
            fex.set_ka_start_free();
            fex.set_ka_val_free();
            fex.set_n_kd_free();
            fex.set_kd_start_free();
            fex.set_kd_vec_free();
            fex.set_kd_val_free();
            fex.set_coeff_free();
            for ( int ind = 0; ind < probSize; ind++) {
                if ( error_sim_h[ind] < E0 ) {
                    bool acc_sub = false;
                    int act_count = 0;
                    int inh_count = 0;
                    int start_ka = *(ka_start+ind);
                    int end_ka = *(ka_start+ind)+*(n_ka+ind);
                    int start_kd = *(kd_start+ind);
                    int end_kd = *(kd_start+ind)+*(n_kd+ind);
                    double r0_ = r0_mc[ind];
                    double ea_ = ea_mc[ind];
                    double d_ = d_mc[ind];
                    output_data(gene_ind, r0_, ea_, d_, start_ka, end_ka, start_kd, end_kd,
                                temp_ka_vec, temp_kd_vec, ka_val_h, kd_val_h, out_files);
                    for ( int k = *(ka_start+ind); k < (*(ka_start+ind)+*(n_ka+ind)); k++) {
                        // Sensitivity for activators
                        if ( ka_val_h[k] > 0.05 ) {
                            act_vec.push_back(temp_ka_vec[k]);
                            acc_sub = true;
                            act_count++;
                        }
                    }
                    for ( int k = *(kd_start+ind); k < (*(kd_start+ind)+*(n_kd+ind)); k++) {
                        // Sensitivity for deactivators
                        if ( kd_val_h[k] > 0.05 ) {
                            inh_vec.push_back(temp_kd_vec[k]);
                            acc_sub = true;
                            inh_count++;
                        }
                    }
                    if (acc_sub) {
                        // Adding accepted candidates
                        acc_subnet++;
                        subnet_size.push_back(act_count+inh_count);
                        cout << "-------- Acceptable subnet number: " << acc_subnet << "  ------------"
                             << endl;
                        cout << "-------- Size of subnetwork: " << act_count+inh_count <<
                             "  ------------" << endl;
                        cout << "-------- Value of error is : " << error_sim_h[ind] << "  ------------"
                             << endl;
                        cout << "-------- Number of attempts made till now: " << subnet_att <<
                             "  ------------" << endl;
                    }
                }
            }
            error_sim_h.clear();
            error_sim_h.shrink_to_fit();
            vector <int>().swap(temp_ka_vec);
            vector <int>().swap(temp_kd_vec);
            vector <double>().swap(temp_ka_val);
            vector <double>().swap(temp_kd_val);
            free(r0);
            free(d);
            free(ea);
            free(n_ka);
            free(n_kd);
            free(ka_start);
            free(kd_start);
            free(ka_vec);
            free(kd_vec);
            free(ka_val);
            free(kd_val);
        }
        stringstream ss;
        ss << (gene_ind + 1);
        string outfile(output_path_c);
        outfile += ss.str();
        outfile += ".out";
//        char out_file[32] = ".out";
//        char index_string[64];
//        sprintf(index_string, "%d", gene_ind+1);
//        strcat(index_string,out_file);
//        char out_path[96] = "../out/N1000/";
//        strcat(out_path,index_string);
        ofstream fileout;
        fileout.open( outfile.c_str(), ios_base::binary|ios_base::app|ios_base::out );
        if (fileout.is_open()) {
            cout << "File open successful!" << endl;
            fileout << "------------ START FOR ACTIVATORS --------------" << endl;
            for ( int ind_con = 0; ind_con < N_gene; ind_con++ ) {
                int act_count = count(act_vec.begin(),act_vec.end(),ind_con);
                fileout << setw(3) << ind_con+1 << "   " << setw(3) << act_count << endl;
            }
            fileout << "-------------- END FOR ACTIVATORS --------------" << endl;
            for ( int ind_con = 0; ind_con < N_gene; ind_con++ ) {
                int inh_count = count(inh_vec.begin(),inh_vec.end(),ind_con);
                fileout << setw(3) << ind_con+1 << "   " << setw(3) << inh_count << endl;
            }
            fileout << "------------ Size of subnetworks START----------- " << endl;
            for ( int ind_con = 0; ind_con < 20; ind_con++ ) {
                int size_count = count(subnet_size.begin(),subnet_size.end(),ind_con);
                fileout << setw(3) << ind_con << "   " << setw(3) << size_count << endl;
            }
            fileout << "------------ Size of subnetworks END----------- " << endl;
            fileout << "------------ Number of acceptable subnetworks for gene " << gene_ind
                    +1 <<
                    " is: " << acc_subnet << " --------------" << endl;
            fileout << "------------ Number of attempts for gene " << gene_ind+1 <<
                    " is: " << subnet_att*probSize << " --------------" << endl;
            fileout.close();
        } else cout << " open() failed" << endl;
        vector <double>().swap(act_vec);
        vector <double>().swap(inh_vec);
        vector <int>().swap(subnet_size);
        cout << "------------ End for gene " << gene_ind+1 << "  ------" << endl;
    }//for (int id_gene = 644; id_gene < 645; id_gene++)
    //
    return 0;
} /* MAIN */

void setup_spline(vector<double>& t_d_in, vector<double> x_d_in,
                  vector<double>& cub_coeff_spline_out, vector<double>& mean_xd_out)
{
    cout << "----Setting up spline co-efficients----" << endl;
    vector <double> deriv_time(N_time_points*N_gene);
    for ( int i = 0 ; i < N_gene ; i++ ) {
        spline_deriv ( i, N_time_points, t_d_in, x_d_in, deriv_time);
    }
    // Set up the co-effients matrix
    spline_coeff ( N_gene, N_time_points, t_d_in, x_d_in, deriv_time,
                   cub_coeff_spline_out);
    // Determine mean, standard deviation, and correlation statistics
    vector<double> sd_xd;
    for ( int ind = 0; ind < N_gene; ind++) {
        //x_spline represented in terms of dt = 1. Used to determine mean, sd, and correlation
        // N=1000 case dt=1 for the data anyway so not necessary
        vector <double> x_spline_temp(N_time_points);
        copy( x_d_in.begin()+ind*N_time_points, x_d_in.begin()+(ind+1)*N_time_points,
              x_spline_temp.begin());
        //calc_spline( x_d, t_d, cub_coeff_spline, ind, x_spline_temp);
        calc_stats( x_spline_temp, mean_xd_out, sd_xd);
        vector<double>().swap(x_spline_temp); // Why?
    }
    cout << "----END spline setup----" << endl;
}

void setup_output(int gene_ind_in, vector<string>& out_files_out) {
    stringstream ss;
    ss << (gene_ind_in + 1);
    string file_prefix(output_path_c);
    file_prefix += ss.str();
    out_files_out.push_back(file_prefix + ".r0");
    out_files_out.push_back(file_prefix + ".ea");
    out_files_out.push_back(file_prefix + ".d");
    out_files_out.push_back(file_prefix + ".nka");
    out_files_out.push_back(file_prefix + ".nkd");
    out_files_out.push_back(file_prefix + ".kavec");
    out_files_out.push_back(file_prefix + ".kdvec");
    out_files_out.push_back(file_prefix + ".kaval");
    out_files_out.push_back(file_prefix + ".kdval");
}

// Error bound
// Max number attempts per gene
// Acceptable subn for each genes

// First loop is which gene (specify start gene end gene)
// Second loop is acceptable subn for each gene and max num for each gene (AND condition)
// Final loop tries to go through generated error values, and sees which are accepted within bounds
// opt contains integrate and MC sim
// fex def is in lsoda/CUlsoda.hpp
