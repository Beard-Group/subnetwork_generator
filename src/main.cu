#include "./common.h"
#include "./io/io.hpp"
#include "./spline/spline.hpp"
#include "./subnetsearch/search.hpp"
#include "./lsoda/cuLsoda.hpp"
#include "./opt/opt.hpp"

using namespace std;
//
int main(void)    
{  
   hipSetDevice(2);
   cout << "----Starting simulation----" << endl; 	
   cout << "----Reading in data----" << endl; 	
   vector <double> t_d(N_time_points);
   vector <double> x_d(N_gene*N_time_points);
   read_data (N_time_points, N_gene, t_d, x_d);
   cout << "----Setting up spline co-efficients----" << endl; 	
   vector <double> deriv_time(N_time_points*N_gene);
   for ( int i = 0 ; i < N_gene ; i++ )
   {
       spline_deriv ( i, N_time_points, t_d, x_d, deriv_time);
   }
   // Set up the co-effients matrix 
   vector <double> cub_coeff_spline (N_gene*N_time_points*4);
   spline_coeff ( N_gene, N_time_points, t_d, x_d, deriv_time, cub_coeff_spline);
   // Determine mean, standard deviation, and correlation statistics
   vector <double> mean_xd, sd_xd;
   for ( int ind = 0; ind < N_gene; ind++)
   { 
       //x_spline represented in terms of dt = 1. Used to determine mean, sd, and correlation
       // N=1000 case dt=1 for the data anyway so not necessary
       vector <double> x_spline_temp(N_time_points);
       copy( x_d.begin()+ind*N_time_points, x_d.begin()+(ind+1)*N_time_points, x_spline_temp.begin());
       //calc_spline( x_d, t_d, cub_coeff_spline, ind, x_spline_temp);
       calc_stats( x_spline_temp, mean_xd, sd_xd);
       vector<double>().swap(x_spline_temp);
   }
   cout << "----END spline setup----" << endl; 	
   //Error
   double E0 = 3.0E-03;
   for (int id_gene = 644; id_gene < 645; id_gene++)
   {
      const int gene_ind = id_gene;
      vector <double> act_vec, inh_vec;
      vector <int> subnet_size;
      bool r0_out = false;
      bool ea_out = false;
      bool d_out = false;
      bool nka_out = false;
      bool nkd_out = false;
      bool kavec_out = false;
      bool kdvec_out = false;
      bool kaval_out = false;
      bool kdval_out = false;
      char r0_out_file[32] = ".r0";
      char ea_out_file[32] = ".ea";
      char d_out_file[32] = ".d";
      char nka_out_file[32] = ".nka";
      char nkd_out_file[32] = ".nkd";
      char kavec_out_file[32] = ".kavec";
      char kdvec_out_file[32] = ".kdvec";
      char kaval_out_file[32] = ".kaval";
      char kdval_out_file[32] = ".kdval";
      char r0_index_string[64];
      char r0_out_path[96] = "../out/N1000/";
      char ea_index_string[64];
      char ea_out_path[96] = "../out/N1000/";
      char d_index_string[64];
      char d_out_path[96] = "../out/N1000/";
      char nka_index_string[64];
      char nka_out_path[96] = "../out/N1000/";
      char nkd_index_string[64];
      char nkd_out_path[96] = "../out/N1000/";
      char kavec_index_string[64];
      char kavec_out_path[96] = "../out/N1000/";
      char kdvec_index_string[64];
      char kdvec_out_path[96] = "../out/N1000/";
      char kaval_index_string[64];
      char kaval_out_path[96] = "../out/N1000/";
      char kdval_index_string[64];
      char kdval_out_path[96] = "../out/N1000/";
      if (!r0_out)
      {
         r0_out = true;
         sprintf(r0_index_string, "%d", gene_ind+1);
         strcat(r0_index_string,r0_out_file);
         strcat(r0_out_path,r0_index_string);
      }
      string out_r0(r0_out_path);
      if (!ea_out)
      {
         ea_out = true;
         sprintf(ea_index_string, "%d", gene_ind+1);
         strcat(ea_index_string,ea_out_file);
         strcat(ea_out_path,ea_index_string);
      }
      string out_ea(ea_out_path);
      if (!d_out)
      {
         d_out = true;
         sprintf(d_index_string, "%d", gene_ind+1);
         strcat(d_index_string,d_out_file);
         strcat(d_out_path,d_index_string);
      }
      string out_d(d_out_path);
      if (!nka_out)
      {
         nka_out = true;
         sprintf(nka_index_string, "%d", gene_ind+1);
         strcat(nka_index_string,nka_out_file);
         strcat(nka_out_path,nka_index_string);
      }
      string out_nka(nka_out_path);
      if (!nkd_out)
      {
         nkd_out = true;
         sprintf(nkd_index_string, "%d", gene_ind+1);
         strcat(nkd_index_string,nkd_out_file);
         strcat(nkd_out_path,nkd_index_string);
      }
      string out_nkd(nkd_out_path);
      if (!kavec_out)
      {
         kavec_out = true;
         sprintf(kavec_index_string, "%d", gene_ind+1);
         strcat(kavec_index_string,kavec_out_file);
         strcat(kavec_out_path,kavec_index_string);
      }
      string out_kavec(kavec_out_path);
      if (!kdvec_out)
      {
         kdvec_out = true;
         sprintf(kdvec_index_string, "%d", gene_ind+1);
         strcat(kdvec_index_string,kdvec_out_file);
         strcat(kdvec_out_path,kdvec_index_string);
      }
      string out_kdvec(kdvec_out_path);
      if (!kaval_out)
      {
         kaval_out = true;
         sprintf(kaval_index_string, "%d", gene_ind+1);
         strcat(kaval_index_string,kaval_out_file);
         strcat(kaval_out_path,kaval_index_string);
      }
      string out_kaval(kaval_out_path);
      if (!kdval_out)
      {
         kdval_out = true;
         sprintf(kdval_index_string, "%d", gene_ind+1);
         strcat(kdval_index_string,kdval_out_file);
         strcat(kdval_out_path,kdval_index_string);
      }
      string out_kdval(kdval_out_path);
      // END DECLARATIONS 
      int acc_subnet = 0;
      int subnet_att = 0;
      cout << "------------ Start for gene " << gene_ind+1 << "  ------" << endl;
      while ( (acc_subnet < N_att) && (subnet_att<50) )
      {
         if (subnet_att <= 500) E0=5.0E-03;
         else E0=7.0E-03;
         cout << "------------ Attempt number " << subnet_att << "  ------" << endl;
         subnet_att++;
         int *n_ka = (int*)malloc(sizeof(int)*probSize);
         int *n_kd = (int*)malloc(sizeof(int)*probSize);
         int *ka_start = (int*)malloc(sizeof(int)*probSize);
         int *kd_start = (int*)malloc(sizeof(int)*probSize);
         int ka_ind = 0;
         int kd_ind = 0;
         vector <int> temp_ka_vec, temp_kd_vec;
         vector <double> temp_ka_val, temp_kd_val;
         srand(time(NULL));
         // generate integers between 50 to 200
         thrust::default_random_engine rng_gen(time(NULL)*rand());
         thrust::uniform_int_distribution<int> u_l(50,200); 
         for ( unsigned int ind = 0; ind < probSize; ind++ )
         {
            int n_ka_temp = 0;
            int n_kd_temp = 0;
            // How to declare kavec_temp and kd_vec_temp
            vector <int> kavec_temp, kdvec_temp;
            vector <double> kaval_temp, kdval_temp;
            for (unsigned int k = 0; k < u_l(rng_gen); k++) 
                random_conn(gene_ind, k+ind, mean_xd, n_ka_temp, n_kd_temp, kavec_temp, kdvec_temp, kaval_temp, kdval_temp);
            *(n_ka+ind) = n_ka_temp;
            *(n_kd+ind) = n_kd_temp;
            *(ka_start+ind) = ka_ind;
            *(kd_start+ind) = kd_ind;
            ka_ind += n_ka_temp;
            kd_ind += n_kd_temp;
            temp_ka_vec.insert(temp_ka_vec.end(),kavec_temp.begin(),kavec_temp.end()); 
            temp_kd_vec.insert(temp_kd_vec.end(),kdvec_temp.begin(),kdvec_temp.end()); 
            temp_ka_val.insert(temp_ka_val.end(),kaval_temp.begin(),kaval_temp.end()); 
            temp_kd_val.insert(temp_kd_val.end(),kdval_temp.begin(),kdval_temp.end());
            vector <int>().swap(kavec_temp);
            vector <int>().swap(kdvec_temp);
            vector <double>().swap(kaval_temp);
            vector <double>().swap(kdval_temp);
         }
         //
         const int size_ka = temp_ka_vec.size();
         const int size_kd = temp_kd_vec.size();
         host_type error_sim_h;
         int *ka_vec = (int*)malloc(sizeof(int)*size_ka);
         double *ka_val = (double*)malloc(sizeof(double)*size_ka);
         int l = 0;
         while ( l < size_ka )
         {
             *(ka_vec+l) = temp_ka_vec[l];
             *(ka_val+l) = temp_ka_val[l];
             l++;
         } 
         l = 0;
         int *kd_vec = (int*)malloc(sizeof(int)*size_kd);
         double *kd_val = (double*)malloc(sizeof(double)*size_kd);
         while ( l < size_kd )
         {
             *(kd_vec+l) = temp_kd_vec[l];
             *(kd_val+l) = temp_kd_val[l];
             l++;
         } 
         double *r0 = (double*)malloc(sizeof(double)*probSize);
         double *d = (double*)malloc(sizeof(double)*probSize);
         double *ea = (double*)malloc(sizeof(double)*probSize);
         srand(time(NULL));
         for ( int i = 0; i < probSize; i++)
         {
               *(r0+i) = mean_xd[gene_ind];
               *(d+i) = 1.0;
               *(ea+i) = 1.0;
         }
         double cub_coeff[N_gene*N_time_points*4];
         for ( int i = 0; i < cub_coeff_spline.size(); i++) cub_coeff[i] = cub_coeff_spline[i];
         myFex fex;
         fex.set_r0(r0);
         fex.set_d(d);
         fex.set_ea(ea);
         fex.set_n_ka(n_ka);
         fex.set_ka_vec(ka_vec,size_ka);
         fex.set_ka_start(ka_start);
         fex.set_ka_val(ka_val,size_ka);
         fex.set_n_kd(n_kd);
         fex.set_kd_start(kd_start);
         fex.set_kd_vec(kd_vec,size_kd);
         fex.set_kd_val(kd_val,size_kd);
         int size_coeff = N_gene*N_time_points*4;
         fex.set_coeff(cub_coeff, size_coeff); 
         myJex jex;
         state_type error_sim_d(probSize);
         // Integrate ODEs
         integrate_lsoda_ode (gene_ind, x_d, t_d, mean_xd[gene_ind], fex, jex, error_sim_d);
         // MC Simulation
         MC_sim(gene_ind, x_d, t_d, mean_xd[gene_ind], n_ka, n_kd, size_ka, size_kd, fex, jex, error_sim_d);
         host_type ka_val_h(size_ka), kd_val_h(size_kd);
         host_type r0_mc(probSize), d_mc(probSize), ea_mc(probSize);
         thrust::device_ptr<double> r0_mc_d, d_mc_d, ea_mc_d;
         fex.get_r0_ptr(r0_mc_d);
         fex.get_ea_ptr(ea_mc_d);
         fex.get_d_ptr(d_mc_d);
         thrust::copy(r0_mc_d, r0_mc_d+probSize, r0_mc.begin()); 
         thrust::copy(ea_mc_d, ea_mc_d+probSize, ea_mc.begin()); 
         thrust::copy(d_mc_d, d_mc_d+probSize, d_mc.begin()); 
         fex.get_kaval_vec(ka_val_h,size_ka);
         fex.get_kdval_vec(kd_val_h,size_kd);
         error_sim_h = error_sim_d;
         fex.set_r0_free();
         fex.set_d_free();
         fex.set_ea_free();
         fex.set_n_ka_free();
         fex.set_ka_vec_free();
         fex.set_ka_start_free();
         fex.set_ka_val_free();
         fex.set_n_kd_free();
         fex.set_kd_start_free();
         fex.set_kd_vec_free();
         fex.set_kd_val_free();
         fex.set_coeff_free(); 
         for ( int ind = 0; ind < probSize; ind++)
         {
             if ( error_sim_h[ind] < E0 ) 
             {
                    bool acc_sub = false; 
                    int act_count = 0;
                    int inh_count = 0;
                    int start_ka = *(ka_start+ind);
                    int end_ka = *(ka_start+ind)+*(n_ka+ind);
                    int start_kd = *(kd_start+ind);
                    int end_kd = *(kd_start+ind)+*(n_kd+ind);
                    double r0_ = r0_mc[ind];
                    double ea_ = ea_mc[ind];
                    double d_ = d_mc[ind];
                    output_data ( gene_ind, r0_, ea_, d_, out_r0, out_ea, out_d, start_ka, end_ka, start_kd, end_kd, temp_ka_vec, temp_kd_vec, ka_val_h, kd_val_h, out_nka, out_nkd, out_kavec, out_kdvec, out_kaval, out_kdval);
                    for ( int k = *(ka_start+ind); k < (*(ka_start+ind)+*(n_ka+ind)); k++) 
                    {
                       if ( ka_val_h[k] > 0.05 ) 
                       {  
                          act_vec.push_back(temp_ka_vec[k]);   
                          acc_sub = true;
                          act_count++;
                       } 
                    }
                    for ( int k = *(kd_start+ind); k < (*(kd_start+ind)+*(n_kd+ind)); k++) 
                    {
                       if ( kd_val_h[k] > 0.05 ) 
                       {   
                          inh_vec.push_back(temp_kd_vec[k]);   
                          acc_sub = true;
                          inh_count++;
                       }
                    }
                    if (acc_sub) 
                    {
                       acc_subnet++;
                       subnet_size.push_back(act_count+inh_count);
                       cout << "-------- Acceptable subnet number: " << acc_subnet << "  ------------" << endl; 
                       cout << "-------- Size of subnetwork: " << act_count+inh_count << "  ------------" << endl; 
                       cout << "-------- Value of error is : " << error_sim_h[ind] << "  ------------" << endl; 
                       cout << "-------- Number of attempts made till now: " << subnet_att << "  ------------" << endl; 
                    }
             } 
         }
         error_sim_h.clear();
         error_sim_h.shrink_to_fit();
         vector <int>().swap(temp_ka_vec);
         vector <int>().swap(temp_kd_vec);
         vector <double>().swap(temp_ka_val);
         vector <double>().swap(temp_kd_val);
         free(r0);
         free(d);
         free(ea);
         free(n_ka);
         free(n_kd);
         free(ka_start);
         free(kd_start);
         free(ka_vec);
         free(kd_vec);
         free(ka_val);
         free(kd_val);
      }
      char out_file[32] = ".out";
      char index_string[64];
      sprintf(index_string, "%d", gene_ind+1);
      strcat(index_string,out_file);
      char out_path[96] = "../out/N1000/";
      strcat(out_path,index_string);
      ofstream fileout;
      fileout.open( out_path, ios_base::binary|ios_base::app|ios_base::out );
      if (fileout.is_open())
      {
         cout << "File open successful!" << endl;
         fileout << "------------ START FOR ACTIVATORS --------------" << endl;
         for ( int ind_con = 0; ind_con < N_gene; ind_con++ )
         {
            int act_count = count(act_vec.begin(),act_vec.end(),ind_con);
            fileout << setw(3) << ind_con+1 << "   " << setw(3) << act_count << endl; 
         }
         fileout << "-------------- END FOR ACTIVATORS --------------" << endl;
         for ( int ind_con = 0; ind_con < N_gene; ind_con++ )
         {
            int inh_count = count(inh_vec.begin(),inh_vec.end(),ind_con); 
            fileout << setw(3) << ind_con+1 << "   " << setw(3) << inh_count << endl; 
         }
         fileout << "------------ Size of subnetworks START----------- " << endl;
         for ( int ind_con = 0; ind_con < 20; ind_con++ )
         {
            int size_count = count(subnet_size.begin(),subnet_size.end(),ind_con); 
            fileout << setw(3) << ind_con << "   " << setw(3) << size_count << endl; 
         }
         fileout << "------------ Size of subnetworks END----------- " << endl;
         fileout << "------------ Number of acceptable subnetworks for gene " << gene_ind+1 <<
          " is: " << acc_subnet << " --------------" << endl;
         fileout << "------------ Number of attempts for gene " << gene_ind+1 <<
          " is: " << subnet_att*probSize << " --------------" << endl;
         fileout.close();
      }
      else cout << " open() failed" << endl;
      vector <double>().swap(act_vec);
      vector <double>().swap(inh_vec);
      vector <int>().swap(subnet_size);
      cout << "------------ End for gene " << gene_ind+1 << "  ------" << endl;
   }
   //
   return 0;
} /* MAIN */
